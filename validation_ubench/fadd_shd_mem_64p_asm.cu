#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime_api.h>
//#include <cutil.h>
#include <hip/hip_runtime.h>

#define GPUJOULE_DIR ""

#define SHARED_MEM_ELEMENTS 1024

int num_blocks;
int num_threads_per_block;
int num_iterations;
int divergence;

float* h_A;
float* h_B;
float* h_C;
float* h_res;
float* d_A;
float* d_B;
float* d_C;
float* d_res;

__global__
//void compute(const float* A, const float* B, const float* C, float* D, int n) {
void compute(float* D, int n, int div, int stride) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    float I1 = tid * 2.0;

    int thread_id = threadIdx.x % 32;

    __shared__ unsigned long long sdata[SHARED_MEM_ELEMENTS];

    __shared__ void **tmp_ptr;

    __shared__ void *arr[SHARED_MEM_ELEMENTS];

    if (threadIdx.x == 0) {
        for (int i = 0; i < SHARED_MEM_ELEMENTS; i++) {
            arr[i] = (void *)&sdata[i];
        }

        for (int i = 0; i < (SHARED_MEM_ELEMENTS - 1); i++) {
            sdata[i] = (unsigned long long) arr[i + 1];
        }

        sdata[SHARED_MEM_ELEMENTS - 1] = (unsigned long long) arr[0];
    }

    __syncthreads();

    tmp_ptr = (void **)(&(arr[(threadIdx.x + stride) % SHARED_MEM_ELEMENTS]));

        double f1, f2, f3;
        f1 = 1.1;
        f2 = 2.5;
    if (thread_id < div) {
/*        __asm volatile (
                ".reg .f32 %r14;\n\t"
                "mov.f32 %r14, 2.2;\n\t"
                );
*/

        for (int k = 0; k < n; k++) {
/*           __asm volatile (
                    "add.rn.f32 %r14, %r11, %r14;\n\t"
                   );
*/           
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + k;
        }
    }
//    __syncthreads();

    //    if ((blockDim.x * blockIdx.x + threadIdx.x) == 0)
    *D = f1 * tid;

//    __syncthreads();
}

void usage() {
    std::cout << "Usage ./binary <num_blocks> <num_threads_per_block> <iterations>" "threads active per warp" << std::endl;
}

int main(int argc, char **argv)
{
    if (argc != 6) {
        usage();
        exit(1);
    }

    int num_blocks = atoi(argv[1]);
    int num_threads_per_block = atoi(argv[2]);
    int iterations = atoi(argv[3]);
    int divergence = atoi(argv[4]);
    int stride = atoi(argv[5]);

//    h_A = new float(2.0);
//    h_B = new float(3.0);
//    h_C = new float(4.0);

//    cudaMalloc((void**)&d_A, sizeof(float));
//    cudaMalloc((void**)&d_B, sizeof(float));
//    cudaMalloc((void**)&d_C, sizeof(float));
    hipMalloc((void**)&d_res, sizeof(float));
  
//    cudaMemcpy(d_A, h_A, sizeof(float), cudaMemcpyHostToDevice);
//    cudaMemcpy(d_B, h_B, sizeof(float), cudaMemcpyHostToDevice);
//    cudaMemcpy(d_C, h_C, sizeof(float), cudaMemcpyHostToDevice);
     
    hipEvent_t start, stop;
    float time;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    std::string cmd = "GPUJOULE_DIR/nvml/example/power_monitor 5 > GPUJOULE_DIR/energy_model_ubench/energy_model_data/combined_inst_validation_data/fadd_shd_mem_64p_asm_power.txt &";
    std::system(cmd.c_str());
    std::system("sleep 5");

    hipEventRecord(start, 0);
    hipProfilerStart();
    
//    compute<<<num_blocks, num_threads_per_block>>>(d_A, d_B, d_C, d_res, iterations);
    compute<<<num_blocks, num_threads_per_block>>>(d_res, iterations, divergence, stride);

    hipProfilerStop();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&time, start, stop);

    std::system("killall power_monitor");
    std::cout << "GPU Elapsed Time = " << time << std::endl;
  
    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipDeviceSynchronize();

    hipMemcpy(h_res, d_res, sizeof(float), hipMemcpyDeviceToHost);

    return 0;
}
