#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime_api.h>
//#include <cutil.h>
#include <hip/hip_runtime.h>
#include <string>

#define GPUJOULE_DIR ""

float* h_A;
float* h_B;
float* h_C;
float* h_res;
float* d_A;
float* d_B;
float* d_C;
float* d_res;

__global__
//void compute(const float* A, const float* B, const float* C, float* D, int n) {
void shared_latency(float* D, int n, int div) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    double I1 = tid * 2.0;

    int thread_id = threadIdx.x % 32;

    if (thread_id < div) {
        __asm volatile (
                " .reg .f64 %r29;\n\t"
                " .reg .f64 %r13;\n\t"
                " .reg .f64 %r14;\n\t"
                " .reg .f64 %r15;\n\t"
                " .reg .f64 %r16;\n\t"
                " .reg .f64 %r17;\n\t"
                " .reg .f64 %r18;\n\t"
                " .reg .f64 %r19;\n\t"
                " .reg .f64 %r20;\n\t"
                " .reg .f64 %r21;\n\t"
                " .reg .f64 %r22;\n\t"
                " .reg .f64 %r23;\n\t"
                " .reg .f64 %r24;\n\t"
                " .reg .f64 %r25;\n\t"
                " .reg .f64 %r26;\n\t"
                " .reg .f64 %r27;\n\t"
                " .reg .f64 %r28;\n\t"
                "mov.f64 %r29, 4.4;\n\t"
                "mov.f64 %r13, %r29;\n\t"
                "mov.f64 %r14, 2.2;\n\t"
                "mov.f64 %r15, 3.3;\n\t"
                "mov.f64 %r16, 1.23;\n\t"
                "mov.f64 %r17, 2.42;\n\t"
                "mov.f64 %r18, 3.34;\n\t"
                "mov.f64 %r19, 5.62;\n\t"
                "mov.f64 %r20, 2.56;\n\t"
                "mov.f64 %r21, 1.56;\n\t"
                "mov.f64 %r22, 2.56;\n\t"
                "mov.f64 %r23, 5.56;\n\t"
                "mov.f64 %r24, 8.56;\n\t"
                "mov.f64 %r25, 3.56;\n\t"
                "mov.f64 %r26, 5.56;\n\t"
                "mov.f64 %r27, 6.56;\n\t"
                "mov.f64 %r28, 0.56;\n\t"

                );
        for (int k = 0; k < n; k++) {
            __asm volatile (
                    "add.rn.f64 %r21, %r29, %r21;\n\t" 
                    "add.rn.f64 %r22, %r29, %r22;\n\t" 
                    "add.rn.f64 %r23, %r29, %r23;\n\t" 
                    "add.rn.f64 %r24, %r29, %r24;\n\t" 
                    "add.rn.f64 %r25, %r29, %r25;\n\t" 
                    "add.rn.f64 %r26, %r29, %r26;\n\t" 
                    "add.rn.f64 %r27, %r29, %r27;\n\t" 
                    "add.rn.f64 %r28, %r29, %r28;\n\t" 
                    "add.rn.f64 %r13, %r29, %r13;\n\t" 
                    "add.rn.f64 %r14, %r29, %r14;\n\t" 
                    "add.rn.f64 %r15, %r29, %r15;\n\t" 
                    "add.rn.f64 %r16, %r29, %r16;\n\t" 
                    "add.rn.f64 %r17, %r29, %r17;\n\t" 
                    "add.rn.f64 %r18, %r29, %r18;\n\t" 
                    "add.rn.f64 %r19, %r29, %r19;\n\t" 
                    "add.rn.f64 %r20, %r29, %r20;\n\t" 
                    "add.rn.f64 %r21, %r29, %r21;\n\t" 
                    "add.rn.f64 %r22, %r29, %r22;\n\t" 
                    "add.rn.f64 %r23, %r29, %r23;\n\t" 
                    "add.rn.f64 %r24, %r29, %r24;\n\t" 
                    "add.rn.f64 %r25, %r29, %r25;\n\t" 
                    "add.rn.f64 %r26, %r29, %r26;\n\t" 
                    "add.rn.f64 %r27, %r29, %r27;\n\t" 
                    "add.rn.f64 %r28, %r29, %r28;\n\t" 
                    "add.rn.f64 %r13, %r29, %r13;\n\t" 
                    "add.rn.f64 %r14, %r29, %r14;\n\t" 
                    "add.rn.f64 %r15, %r29, %r15;\n\t" 
                    "add.rn.f64 %r16, %r29, %r16;\n\t" 
                    "add.rn.f64 %r17, %r29, %r17;\n\t" 
                    "add.rn.f64 %r18, %r29, %r18;\n\t" 
                    "add.rn.f64 %r19, %r29, %r19;\n\t" 
                    "add.rn.f64 %r20, %r29, %r20;\n\t" 
                    "add.rn.f64 %r21, %r29, %r21;\n\t" 
                    "add.rn.f64 %r22, %r29, %r22;\n\t" 
                    "add.rn.f64 %r23, %r29, %r23;\n\t" 
                    "add.rn.f64 %r24, %r29, %r24;\n\t" 
                    "add.rn.f64 %r25, %r29, %r25;\n\t" 
                    "add.rn.f64 %r26, %r29, %r26;\n\t" 
                    "add.rn.f64 %r27, %r29, %r27;\n\t" 
                    "add.rn.f64 %r28, %r29, %r28;\n\t" 
                    "add.rn.f64 %r13, %r29, %r13;\n\t" 
                    "add.rn.f64 %r14, %r29, %r14;\n\t" 
                    "add.rn.f64 %r15, %r29, %r15;\n\t" 
                    "add.rn.f64 %r16, %r29, %r16;\n\t" 
                    "add.rn.f64 %r17, %r29, %r17;\n\t" 
                    "add.rn.f64 %r18, %r29, %r18;\n\t" 
                    "add.rn.f64 %r19, %r29, %r19;\n\t" 
                    "add.rn.f64 %r20, %r29, %r20;\n\t" 
                    "add.rn.f64 %r21, %r29, %r21;\n\t" 
                    "add.rn.f64 %r22, %r29, %r22;\n\t" 
                    "add.rn.f64 %r23, %r29, %r23;\n\t" 
                    "add.rn.f64 %r24, %r29, %r24;\n\t" 
                    "add.rn.f64 %r25, %r29, %r25;\n\t" 
                    "add.rn.f64 %r26, %r29, %r26;\n\t" 
                    "add.rn.f64 %r27, %r29, %r27;\n\t" 
                    "add.rn.f64 %r28, %r29, %r28;\n\t" 
                    "add.rn.f64 %r13, %r29, %r13;\n\t" 
                    "add.rn.f64 %r14, %r29, %r14;\n\t" 
                    "add.rn.f64 %r15, %r29, %r15;\n\t" 
                    "add.rn.f64 %r16, %r29, %r16;\n\t" 
                    "add.rn.f64 %r17, %r29, %r17;\n\t" 
                    "add.rn.f64 %r18, %r29, %r18;\n\t" 
                    "add.rn.f64 %r19, %r29, %r19;\n\t" 
                    "add.rn.f64 %r20, %r29, %r20;\n\t" 
                    "add.rn.f64 %r21, %r29, %r21;\n\t" 
                    "add.rn.f64 %r22, %r29, %r22;\n\t" 
                    "add.rn.f64 %r23, %r29, %r23;\n\t" 
                    "add.rn.f64 %r24, %r29, %r24;\n\t" 
                    "add.rn.f64 %r25, %r29, %r25;\n\t" 
                    "add.rn.f64 %r26, %r29, %r26;\n\t" 
                    "add.rn.f64 %r27, %r29, %r27;\n\t" 
                    "add.rn.f64 %r28, %r29, %r28;\n\t" 
                    "add.rn.f64 %r13, %r29, %r13;\n\t" 
                    "add.rn.f64 %r14, %r29, %r14;\n\t" 
                    "add.rn.f64 %r15, %r29, %r15;\n\t" 
                    "add.rn.f64 %r16, %r29, %r16;\n\t" 
                    "add.rn.f64 %r17, %r29, %r17;\n\t" 
                    "add.rn.f64 %r18, %r29, %r18;\n\t" 
                    "add.rn.f64 %r19, %r29, %r19;\n\t" 
                    "add.rn.f64 %r20, %r29, %r20;\n\t" 
                    "add.rn.f64 %r21, %r29, %r21;\n\t" 
                    "add.rn.f64 %r22, %r29, %r22;\n\t" 
                    "add.rn.f64 %r23, %r29, %r23;\n\t" 
                    "add.rn.f64 %r24, %r29, %r24;\n\t" 
                    "add.rn.f64 %r25, %r29, %r25;\n\t" 
                    "add.rn.f64 %r26, %r29, %r26;\n\t" 
                    "add.rn.f64 %r27, %r29, %r27;\n\t" 
                    "add.rn.f64 %r28, %r29, %r28;\n\t" 
                    "add.rn.f64 %r13, %r29, %r13;\n\t" 
                    "add.rn.f64 %r14, %r29, %r14;\n\t" 
                    "add.rn.f64 %r15, %r29, %r15;\n\t" 
                    "add.rn.f64 %r16, %r29, %r16;\n\t" 
                    "add.rn.f64 %r17, %r29, %r17;\n\t" 
                    "add.rn.f64 %r18, %r29, %r18;\n\t" 
                    "add.rn.f64 %r19, %r29, %r19;\n\t" 
                    "add.rn.f64 %r20, %r29, %r20;\n\t" 
                    "add.rn.f64 %r21, %r29, %r21;\n\t" 
                    "add.rn.f64 %r22, %r29, %r22;\n\t" 
                    "add.rn.f64 %r23, %r29, %r23;\n\t" 
                    "add.rn.f64 %r24, %r29, %r24;\n\t" 
                    "add.rn.f64 %r25, %r29, %r25;\n\t" 
                    "add.rn.f64 %r26, %r29, %r26;\n\t" 
                    "add.rn.f64 %r27, %r29, %r27;\n\t" 
                    "add.rn.f64 %r28, %r29, %r28;\n\t" 
                    "add.rn.f64 %r13, %r29, %r13;\n\t" 
                    "add.rn.f64 %r14, %r29, %r14;\n\t" 
                    "add.rn.f64 %r15, %r29, %r15;\n\t" 
                    "add.rn.f64 %r16, %r29, %r16;\n\t" 
                    "add.rn.f64 %r17, %r29, %r17;\n\t" 
                    "add.rn.f64 %r18, %r29, %r18;\n\t" 
                    "add.rn.f64 %r19, %r29, %r19;\n\t" 
                    "add.rn.f64 %r20, %r29, %r20;\n\t" 
                    "add.rn.f64 %r21, %r29, %r21;\n\t" 
                    "add.rn.f64 %r22, %r29, %r22;\n\t" 
                    "add.rn.f64 %r23, %r29, %r23;\n\t" 
                    "add.rn.f64 %r24, %r29, %r24;\n\t" 
                    "add.rn.f64 %r25, %r29, %r25;\n\t" 
                    "add.rn.f64 %r26, %r29, %r26;\n\t" 
                    "add.rn.f64 %r27, %r29, %r27;\n\t" 
                    "add.rn.f64 %r28, %r29, %r28;\n\t" 
                    "add.rn.f64 %r13, %r29, %r13;\n\t" 
                    "add.rn.f64 %r14, %r29, %r14;\n\t" 
                    "add.rn.f64 %r15, %r29, %r15;\n\t" 
                    "add.rn.f64 %r16, %r29, %r16;\n\t" 
                    "add.rn.f64 %r17, %r29, %r17;\n\t" 
                    "add.rn.f64 %r18, %r29, %r18;\n\t" 
                    "add.rn.f64 %r19, %r29, %r19;\n\t" 
                    "add.rn.f64 %r20, %r29, %r20;\n\t" 
                    "add.rn.f64 %r21, %r29, %r21;\n\t" 
                    "add.rn.f64 %r22, %r29, %r22;\n\t" 
                    "add.rn.f64 %r23, %r29, %r23;\n\t" 
                    "add.rn.f64 %r24, %r29, %r24;\n\t" 
                    "add.rn.f64 %r25, %r29, %r25;\n\t" 
                    "add.rn.f64 %r26, %r29, %r26;\n\t" 
                    "add.rn.f64 %r27, %r29, %r27;\n\t" 
                    "add.rn.f64 %r28, %r29, %r28;\n\t" 
                    "add.rn.f64 %r13, %r29, %r13;\n\t" 
                    "add.rn.f64 %r14, %r29, %r14;\n\t" 
                    "add.rn.f64 %r15, %r29, %r15;\n\t" 
                    "add.rn.f64 %r16, %r29, %r16;\n\t" 
                    "add.rn.f64 %r17, %r29, %r17;\n\t" 
                    "add.rn.f64 %r18, %r29, %r18;\n\t" 
                    "add.rn.f64 %r19, %r29, %r19;\n\t" 
                    "add.rn.f64 %r20, %r29, %r20;\n\t" 
                    "add.rn.f64 %r21, %r29, %r21;\n\t" 
                    "add.rn.f64 %r22, %r29, %r22;\n\t" 
                    "add.rn.f64 %r23, %r29, %r23;\n\t" 
                    "add.rn.f64 %r24, %r29, %r24;\n\t" 
                    "add.rn.f64 %r25, %r29, %r25;\n\t" 
                    "add.rn.f64 %r26, %r29, %r26;\n\t" 
                    "add.rn.f64 %r27, %r29, %r27;\n\t" 
                    "add.rn.f64 %r28, %r29, %r28;\n\t" 
                    "add.rn.f64 %r13, %r29, %r13;\n\t" 
                    "add.rn.f64 %r14, %r29, %r14;\n\t" 
                    "add.rn.f64 %r15, %r29, %r15;\n\t" 
                    "add.rn.f64 %r16, %r29, %r16;\n\t" 
                    "add.rn.f64 %r17, %r29, %r17;\n\t" 
                    "add.rn.f64 %r18, %r29, %r18;\n\t" 
                    "add.rn.f64 %r19, %r29, %r19;\n\t" 
                    "add.rn.f64 %r20, %r29, %r20;\n\t" 
                    "add.rn.f64 %r21, %r29, %r21;\n\t" 
                    "add.rn.f64 %r22, %r29, %r22;\n\t" 
                    "add.rn.f64 %r23, %r29, %r23;\n\t" 
                    "add.rn.f64 %r24, %r29, %r24;\n\t" 
                    "add.rn.f64 %r25, %r29, %r25;\n\t" 
                    "add.rn.f64 %r26, %r29, %r26;\n\t" 
                    "add.rn.f64 %r27, %r29, %r27;\n\t" 
                    "add.rn.f64 %r28, %r29, %r28;\n\t" 
                    "add.rn.f64 %r13, %r29, %r13;\n\t" 
                    "add.rn.f64 %r14, %r29, %r14;\n\t" 
                    "add.rn.f64 %r15, %r29, %r15;\n\t" 
                    "add.rn.f64 %r16, %r29, %r16;\n\t" 
                    "add.rn.f64 %r17, %r29, %r17;\n\t" 
                    "add.rn.f64 %r18, %r29, %r18;\n\t" 
                    "add.rn.f64 %r19, %r29, %r19;\n\t" 
                    "add.rn.f64 %r20, %r29, %r20;\n\t" 
                    "add.rn.f64 %r21, %r29, %r21;\n\t" 
                    "add.rn.f64 %r22, %r29, %r22;\n\t" 
                    "add.rn.f64 %r23, %r29, %r23;\n\t" 
                    "add.rn.f64 %r24, %r29, %r24;\n\t" 
                    "add.rn.f64 %r25, %r29, %r25;\n\t" 
                    "add.rn.f64 %r26, %r29, %r26;\n\t" 
                    "add.rn.f64 %r27, %r29, %r27;\n\t" 
                    "add.rn.f64 %r28, %r29, %r28;\n\t" 
                    "add.rn.f64 %r13, %r29, %r13;\n\t" 
                    "add.rn.f64 %r14, %r29, %r14;\n\t" 
                    "add.rn.f64 %r15, %r29, %r15;\n\t" 
                    "add.rn.f64 %r16, %r29, %r16;\n\t" 
                    "add.rn.f64 %r17, %r29, %r17;\n\t" 
                    "add.rn.f64 %r18, %r29, %r18;\n\t" 
                    "add.rn.f64 %r19, %r29, %r19;\n\t" 
                    "add.rn.f64 %r20, %r29, %r20;\n\t" 
                    "add.rn.f64 %r21, %r29, %r21;\n\t" 
                    "add.rn.f64 %r22, %r29, %r22;\n\t" 
                    "add.rn.f64 %r23, %r29, %r23;\n\t" 
                    "add.rn.f64 %r24, %r29, %r24;\n\t" 
                    "add.rn.f64 %r25, %r29, %r25;\n\t" 
                    "add.rn.f64 %r26, %r29, %r26;\n\t" 
                    "add.rn.f64 %r27, %r29, %r27;\n\t" 
                    "add.rn.f64 %r28, %r29, %r28;\n\t" 
                    );
        }
   
//        double temp; 
//        float output = 0.0;
//        asm("add.rn.f64 %0, r13, r14" : "=d"(temp));
//        asm("cvt.rn.f32.f64 %0, %1" : "=f"(output) : "d"(temp));
//        printf("%lf \n", output);
    }
    __syncthreads();

    //    if ((blockDim.x * blockIdx.x + threadIdx.x) == 0)
    *D = I1;

    __syncthreads();
}

void usage() {
    std::cout << "Usage ./binary <num_blocks> <num_threads_per_block> <iterations>" "threads active per warp" << std::endl;
}

int main(int argc, char **argv)
{
    if (argc != 6) {
        usage();
        exit(1);
    }

    int num_blocks = atoi(argv[1]);
    int num_threads_per_block = atoi(argv[2]);
    int iterations = atoi(argv[3]);
    int divergence = atoi(argv[4]);
    int stride = atoi(argv[5]);

//    h_A = new float(2.0);
//    h_B = new float(3.0);
//    h_C = new float(4.0);

//    cudaMalloc((void**)&d_A, sizeof(float));
//    cudaMalloc((void**)&d_B, sizeof(float));
//    cudaMalloc((void**)&d_C, sizeof(float));
    hipMalloc((void**)&d_res, sizeof(double));
  
//    cudaMemcpy(d_A, h_A, sizeof(float), cudaMemcpyHostToDevice);
//    cudaMemcpy(d_B, h_B, sizeof(float), cudaMemcpyHostToDevice);
//    cudaMemcpy(d_C, h_C, sizeof(float), cudaMemcpyHostToDevice);
     
    hipEvent_t start, stop;
    float time;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    std::string cmd = "GPUJOULE_DIR/nvml/example/power_monitor 5 > GPUJOULE_DIR/energy_model_ubench/energy_model_data/data_movement_energy/l2_cache/fadd_l2d_100_0_64p_asm_power.txt &";
    std::system(cmd.c_str());
    std::system("sleep 5");
   
    hipEventRecord(start, 0);
    hipProfilerStart();
    
//    compute<<<num_blocks, num_threads_per_block>>>(d_A, d_B, d_C, d_res, iterations);
    shared_latency<<<num_blocks, num_threads_per_block>>>(d_res, iterations, divergence);

    hipDeviceSynchronize();
    hipProfilerStop();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&time, start, stop);

    std::system("killall power_monitor");
    std::cout << time << std::endl;
  
    hipEventDestroy(start);
    hipEventDestroy(stop);


    hipMemcpy(h_res, d_res, sizeof(double), hipMemcpyDeviceToHost);

    return 0;
}
