#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime_api.h>
//#include <cutil.h>
#include <hip/hip_runtime.h>
#include <string>

#define GPUJOULE_DIR ""

#define SHARED_MEM_ELEMENTS 1024
#define GLOBAL_MEM_ELEMENTS 4096

int num_blocks;
int num_threads_per_block;
int num_iterations;
int divergence;

__global__ void shared_latency (unsigned long long ** my_ptr_array, unsigned long long * my_array, int array_length, int iterations, unsigned long long * duration, int stride, int divergence, int num_blocks_k, int num_threads_per_block_k) {

//    unsigned long long int start_time, end_time;
    unsigned long long int sum_time = 0;
    int i, k;

    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    int warp_thread_id = threadIdx.x % 32;
    __shared__ unsigned long long sdata[SHARED_MEM_ELEMENTS];

    __shared__ void **tmp_ptr;

    __shared__ void *arr[SHARED_MEM_ELEMENTS];
    
    if (threadIdx.x == 0) {
        for (i=0; i < SHARED_MEM_ELEMENTS; i++) {
            arr[i] = (void *)&sdata[i];
        }
        for (i=0; i < (SHARED_MEM_ELEMENTS - 1); i++) {
            sdata[i] = (unsigned long long)arr[i+1];
        }
        sdata[SHARED_MEM_ELEMENTS - 1] = (unsigned long long) arr[0];
    }

    __syncthreads();

    tmp_ptr = (void **)(&(arr[(threadIdx.x + stride)%SHARED_MEM_ELEMENTS]));

        double f1, f2, f3;
        f1 = 1.1;
        f2 = 2.5;
    if (warp_thread_id < divergence) {

        for (int l = 0; l < iterations; l++) {
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
        }
    }
//    __syncthreads();

    //    if ((blockDim.x * blockIdx.x + threadIdx.x) == 0)
    duration[tid] = (unsigned long long)(*tmp_ptr) + (f1 * tid);

//    __syncthreads();
}

void usage() {
    std::cout << "Usage ./binary <num_blocks> <num_threads_per_block> <iterations>" "threads active per warp" << std::endl;
}

void parametric_measure_shared(int N, int iterations, int stride) {

    hipProfilerStop();
    int i;
    unsigned long long int * h_a;
    unsigned long long int * d_a;

    unsigned long long ** h_ptr_a;
    unsigned long long ** d_ptr_a;

    unsigned long long * duration;
    unsigned long long * latency;

    hipError_t error_id;

    /* allocate array on CPU */
    h_a = (unsigned long long *)malloc(sizeof(unsigned long long int) * N);

    h_ptr_a = (unsigned long long **)malloc(sizeof(unsigned long long int*)*N);

    latency = (unsigned long long *)malloc(sizeof(unsigned long long) * num_threads_per_block * num_blocks);

    /* initialize array elements on CPU */


    for (i = 0; i < N; i++) {
        h_ptr_a[i] = (unsigned long long *)&h_a[i];
    }
    for (i = 0; i < N; i++) {
        h_a[i] = (unsigned long long)h_ptr_a[(i + 1 + stride) % N];	
    }

    /* allocate arrays on GPU */
    hipMalloc ((void **) &d_a, sizeof(unsigned long long int) * N );
    hipMalloc ((void **) &d_ptr_a, sizeof(unsigned long long int*) * N );
    hipMalloc ((void **) &duration, sizeof(unsigned long long) * num_threads_per_block * num_blocks);

    hipDeviceSynchronize ();
    error_id = hipGetLastError();
    if (error_id != hipSuccess) {
        printf("Error 1 is %s\n", hipGetErrorString(error_id));
    }

    /* copy array elements from CPU to GPU */
    hipMemcpy((void *)d_a, (void *)h_a, sizeof(unsigned long long int) * N, hipMemcpyHostToDevice);
    hipMemcpy((void *)d_ptr_a, (void *)h_ptr_a, sizeof(unsigned long long int *) * N, hipMemcpyHostToDevice);
    hipMemcpy((void *)duration, (void *)latency, sizeof(unsigned long long) * num_threads_per_block * num_blocks, hipMemcpyHostToDevice);

    hipDeviceSynchronize ();

    error_id = hipGetLastError();
    if (error_id != hipSuccess) {
        printf("Error 2 is %s\n", hipGetErrorString(error_id));
    }

//    init_memory <<<1, 1>>>(d_ptr_a, d_a, stride, num_blocks, num_threads_per_block);
//    hipDeviceSynchronize();

    /* launch kernel*/
    //dim3 Db = dim3(13);
    //dim3 Dg = dim3(768,1,1);

    //printf("Launch kernel with parameters: %d, N: %d, stride: %d\n", iterations, N, stride); 
    //	int sharedMemSize =  sizeof(unsigned long long int) * N ;

    hipEvent_t start, stop;
    float time;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    std::string cmd = "GPUJOULE_DIR/nvml/example/power_monitor 5 > GPUJOULE_DIR/energy_model_ubench/energy_model_data/data_movement_energy/shd_mem/fadd_shd_0_100_64p_asm_power.txt &";
    std::system(cmd.c_str());
    std::system("sleep 5");
   
    hipEventRecord(start, 0);
    hipProfilerStart();

    hipFuncSetCacheConfig(reinterpret_cast<const void*>(shared_latency), hipFuncCachePreferL1);
    //shared_latency <<<Dg, Db, sharedMemSize>>>(d_a, N, iterations, duration);
    //shared_latency <<<num_blocks, num_threads_per_block, sharedMemSize>>>(d_a, N, num_iterations, duration, stride, divergence);
    shared_latency <<<num_blocks, num_threads_per_block>>>(d_ptr_a, d_a, N, num_iterations, duration, stride, divergence, num_blocks, num_threads_per_block);

    hipDeviceSynchronize();
    ///hipDeviceSynchronize ();

    hipProfilerStop();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&time, start, stop);

    std::system("killall power_monitor");

    error_id = hipGetLastError();
    if (error_id != hipSuccess) {
        printf("Error 3 is %s\n", hipGetErrorString(error_id));
    }

    /* copy results from GPU to CPU */

    hipMemcpy((void *)h_a, (void *)d_a, sizeof(unsigned long long int) * N, hipMemcpyDeviceToHost);
    hipMemcpy((void *)latency, (void *)duration, sizeof(unsigned long long) * num_threads_per_block * num_blocks, hipMemcpyDeviceToHost);

    hipDeviceSynchronize ();

    /* print results*/


    unsigned long long max_dur = latency[0];
    unsigned long long min_dur = latency[0];
    unsigned long long avg_lat = latency[0];
    for (int i = 1; i < num_threads_per_block * num_blocks; i++) {
        avg_lat += latency[i];
        if (latency[i] > max_dur) {
            max_dur = latency[i];
        } else if (latency[i] < min_dur) {
            min_dur = latency[i];
        }
    }


    //	printf("  %d, %f, %f, %f, %f\n",stride,(double)(avg_lat/(num_threads_per_block * num_blocks * 256.0 *num_iterations)), (double)(min_dur/(256.0 * num_iterations)), (double)(max_dur/(256.0 * num_iterations)), time);

    printf("%f\n", time);


    /* free memory on GPU */
    hipFree(d_a);
    hipFree(d_ptr_a);
    hipFree(duration);
    hipDeviceSynchronize ();

    /*free memory on CPU */
    free(h_a);
    free(h_ptr_a);
    free(latency);


}


int main(int argc, char **argv)
{
    int N;

    if (argc != 6) {
        usage();
        exit(1);
    }

    num_blocks = atoi(argv[1]);
    num_threads_per_block = atoi(argv[2]);
    num_iterations = atoi(argv[3]);
    divergence = atoi(argv[4]);
    int stride = atoi(argv[5]);

    N = GLOBAL_MEM_ELEMENTS;
    parametric_measure_shared(N, 10, stride);

    return 0;
}
