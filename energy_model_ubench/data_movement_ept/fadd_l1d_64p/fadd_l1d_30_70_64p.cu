#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime_api.h>
//#include <cutil.h>
#include <hip/hip_runtime.h>
#include <string>

#define GPUJOULE_DIR ""

#define SHARED_MEM_ELEMENTS 1024
#define GLOBAL_MEM_ELEMENTS 4096

int num_blocks;
int num_threads_per_block;
int num_iterations;
int divergence;

float* h_A;
float* h_B;
float* h_C;
float* h_res;
float* d_A;
float* d_B;
float* d_C;
float* d_res;

__global__ void init_memory (unsigned long long ** my_ptr_array, unsigned long long * my_array, int stride, int num_blocks_k, int num_threads_per_block_k) {

    int block_id;
    int warp_id;
    int i;

    int index;
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    void **ptr_array = (void **)my_ptr_array;
    unsigned long long *array = (unsigned long long *)my_array;

    if (tid == 0) {
//        int elements_per_block = GLOBAL_MEM_ELEMENTS / num_blocks_k;
        int num_warps_per_block = num_threads_per_block_k / 32;
        //int elements_per_warp = elements_per_block / num_warps_per_block;
        int elements_per_warp = GLOBAL_MEM_ELEMENTS / num_warps_per_block;
        
//        for (block_id = 0; block_id < num_blocks_k; block_id++) {
            for (warp_id = 0; warp_id < num_warps_per_block; warp_id++) {
                for (i = 0; i < elements_per_warp; i++) {
                    //index = (block_id * elements_per_block) + (warp_id * elements_per_warp);
                    index = (warp_id * elements_per_warp);
                    ptr_array[index + i] = (void*)&array[(index + ((i + 16) % elements_per_warp))];
                }
            }

/*        for (i = 0; i < GLOBAL_MEM_ELEMENTS; i++) {
            ptr_array[i] = (void*)&array[(i + 32)%GLOBAL_MEM_ELEMENTS];
        }
*/
        for (i = 0; i < GLOBAL_MEM_ELEMENTS; i++) {
            //array[i] = (unsigned long long)ptr_array[(i+stride)%GLOBAL_MEM_ELEMENTS];
            array[i] = (unsigned long long)ptr_array[i];
        }
    }

    __syncthreads();

}

__global__ void shared_latency (unsigned long long ** my_ptr_array, unsigned long long * my_array, int array_length, int iterations, unsigned long long * duration, int stride, int divergence, int num_blocks_k, int num_threads_per_block_k) {

//    unsigned long long int start_time, end_time;
    unsigned long long int sum_time = 0;
    int i, k;

    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    int block_id = blockIdx.x;
    int warp_id = threadIdx.x / 32;
    int warp_thread_id = threadIdx.x % 32;

//    int elements_per_block = GLOBAL_MEM_ELEMENTS / num_blocks_k;
    int num_warps_per_block = num_threads_per_block_k / 32;
//    int elements_per_warp = elements_per_block / num_warps_per_block;
    int elements_per_warp = GLOBAL_MEM_ELEMENTS / num_warps_per_block;

    //int index1 = (block_id * elements_per_block) + (warp_id * elements_per_warp) + warp_thread_id;
    int index1 = (warp_id * elements_per_warp) + warp_thread_id;

    void **ptr_array = (void **)my_ptr_array;
    unsigned long long int *array = (unsigned long long int *)my_array;

    void **tmp_ptr; 
    
    //tmp_ptr = (void *)sdata;
    //tmp_ptr = (void **)(&(ptr_array[(threadIdx.x * stride)%GLOBAL_MEM_ELEMENTS]));
    //tmp_ptr = (void **)(&(ptr_array[(tid * stride)%GLOBAL_MEM_ELEMENTS]));
    //tmp_ptr = (void **)(&(ptr_array[index1]));
    tmp_ptr = (void **)(&(array[index1]));

        double f1, f2, f3;
        f1 = 1.1;
        f2 = 2.5;
    if (warp_thread_id < divergence) {
/*        __asm volatile (
                ".reg .f32 %r14;\n\t"
                "mov.f32 %r14, 2.2;\n\t"
                );
*/

        for (int l = 0; l < iterations; l++) {
            f1 = f1 + (unsigned long long)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + (unsigned long long)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + (unsigned long long)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + (unsigned long long)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + (unsigned long long)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + (unsigned long long)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + (unsigned long long)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + (unsigned long long)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + (unsigned long long)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + (unsigned long long)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + (unsigned long long)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + (unsigned long long)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + (unsigned long long)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + (unsigned long long)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + (unsigned long long)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + (unsigned long long)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + (unsigned long long)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + (unsigned long long)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + (unsigned long long)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + (unsigned long long)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + (unsigned long long)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + (unsigned long long)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + (unsigned long long)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + (unsigned long long)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + (unsigned long long)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + (unsigned long long)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + (unsigned long long)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + (unsigned long long)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + (unsigned long long)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + (unsigned long long)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + (unsigned long long)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + (unsigned long long)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + (unsigned long long)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + (unsigned long long)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + (unsigned long long)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + (unsigned long long)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + (unsigned long long)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + (unsigned long long)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + (unsigned long long)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + (unsigned long long)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + (unsigned long long)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + (unsigned long long)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + (unsigned long long)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + (unsigned long long)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + (unsigned long long)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + (unsigned long long)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + (unsigned long long)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + (unsigned long long)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + (unsigned long long)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + (unsigned long long)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + (unsigned long long)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + (unsigned long long)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + (unsigned long long)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + (unsigned long long)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + (unsigned long long)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + (unsigned long long)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + (unsigned long long)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + (unsigned long long)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + (unsigned long long)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
            f1 = f1 + (unsigned long long)(*tmp_ptr);
            tmp_ptr = (void**)(*tmp_ptr);
        }
    }
//    __syncthreads();

    //    if ((blockDim.x * blockIdx.x + threadIdx.x) == 0)
    duration[tid] = (unsigned long long)(*tmp_ptr) + (f1 * tid);

//    __syncthreads();
}

void usage() {
    std::cout << "Usage ./binary <num_blocks> <num_threads_per_block> <iterations>" "threads active per warp" << std::endl;
}

void parametric_measure_shared(int N, int iterations, int stride) {

    hipProfilerStop();
    int i;
    unsigned long long int * h_a;
    unsigned long long int * d_a;

    unsigned long long ** h_ptr_a;
    unsigned long long ** d_ptr_a;

    unsigned long long * duration;
    unsigned long long * latency;

    hipError_t error_id;

    /* allocate array on CPU */
    h_a = (unsigned long long *)malloc(sizeof(unsigned long long int) * N);

    h_ptr_a = (unsigned long long **)malloc(sizeof(unsigned long long int*)*N);

    latency = (unsigned long long *)malloc(sizeof(unsigned long long) * num_threads_per_block * num_blocks);

    /* initialize array elements on CPU */


    for (i = 0; i < N; i++) {
        h_ptr_a[i] = (unsigned long long *)&h_a[i];
    }
    for (i = 0; i < N; i++) {
        h_a[i] = (unsigned long long)h_ptr_a[(i + 1 + stride) % N];	
    }

    /* allocate arrays on GPU */
    hipMalloc ((void **) &d_a, sizeof(unsigned long long int) * N );
    hipMalloc ((void **) &d_ptr_a, sizeof(unsigned long long int*) * N );
    hipMalloc ((void **) &duration, sizeof(unsigned long long) * num_threads_per_block * num_blocks);

    hipDeviceSynchronize ();
    error_id = hipGetLastError();
    if (error_id != hipSuccess) {
        printf("Error 1 is %s\n", hipGetErrorString(error_id));
    }

    /* copy array elements from CPU to GPU */
    hipMemcpy((void *)d_a, (void *)h_a, sizeof(unsigned long long int) * N, hipMemcpyHostToDevice);
    hipMemcpy((void *)d_ptr_a, (void *)h_ptr_a, sizeof(unsigned long long int *) * N, hipMemcpyHostToDevice);
    hipMemcpy((void *)duration, (void *)latency, sizeof(unsigned long long) * num_threads_per_block * num_blocks, hipMemcpyHostToDevice);

    hipDeviceSynchronize ();

    error_id = hipGetLastError();
    if (error_id != hipSuccess) {
        printf("Error 2 is %s\n", hipGetErrorString(error_id));
    }

    init_memory <<<1, 1>>>(d_ptr_a, d_a, stride, num_blocks, num_threads_per_block);
    hipDeviceSynchronize();

    /* launch kernel*/
    //dim3 Db = dim3(13);
    //dim3 Dg = dim3(768,1,1);

    //printf("Launch kernel with parameters: %d, N: %d, stride: %d\n", iterations, N, stride); 
    //	int sharedMemSize =  sizeof(unsigned long long int) * N ;

    hipEvent_t start, stop;
    float time;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    std::string cmd = "GPUJOULE_DIR/nvml/example/power_monitor 5 > GPUJOULE_DIR/energy_model_ubench/energy_model_data/data_movement_energy/l1_cache/fadd_l1d_30_70_64p_asm_power.txt &";
    std::system(cmd.c_str());
    std::system("sleep 5");

    hipEventRecord(start, 0);
    hipProfilerStart();

    hipFuncSetCacheConfig(reinterpret_cast<const void*>(shared_latency), hipFuncCachePreferL1);
    //shared_latency <<<Dg, Db, sharedMemSize>>>(d_a, N, iterations, duration);
    //shared_latency <<<num_blocks, num_threads_per_block, sharedMemSize>>>(d_a, N, num_iterations, duration, stride, divergence);
    shared_latency <<<num_blocks, num_threads_per_block>>>(d_ptr_a, d_a, N, num_iterations, duration, stride, divergence, num_blocks, num_threads_per_block);

    hipDeviceSynchronize();
    ///hipDeviceSynchronize ();

    hipProfilerStop();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&time, start, stop);

    std::system("killall power_monitor");

    error_id = hipGetLastError();
    if (error_id != hipSuccess) {
        printf("Error 3 is %s\n", hipGetErrorString(error_id));
    }

    /* copy results from GPU to CPU */

    hipMemcpy((void *)h_a, (void *)d_a, sizeof(unsigned long long int) * N, hipMemcpyDeviceToHost);
    hipMemcpy((void *)latency, (void *)duration, sizeof(unsigned long long) * num_threads_per_block * num_blocks, hipMemcpyDeviceToHost);

    hipDeviceSynchronize ();

    /* print results*/


    unsigned long long max_dur = latency[0];
    unsigned long long min_dur = latency[0];
    unsigned long long avg_lat = latency[0];
    for (int i = 1; i < num_threads_per_block * num_blocks; i++) {
        avg_lat += latency[i];
        if (latency[i] > max_dur) {
            max_dur = latency[i];
        } else if (latency[i] < min_dur) {
            min_dur = latency[i];
        }
    }


    //	printf("  %d, %f, %f, %f, %f\n",stride,(double)(avg_lat/(num_threads_per_block * num_blocks * 256.0 *num_iterations)), (double)(min_dur/(256.0 * num_iterations)), (double)(max_dur/(256.0 * num_iterations)), time);

    printf("%f\n", time);


    /* free memory on GPU */
    hipFree(d_a);
    hipFree(d_ptr_a);
    hipFree(duration);
    hipDeviceSynchronize ();

    /*free memory on CPU */
    free(h_a);
    free(h_ptr_a);
    free(latency);


}


int main(int argc, char **argv)
{
    int N;

    if (argc != 6) {
        usage();
        exit(1);
    }

    num_blocks = atoi(argv[1]);
    num_threads_per_block = atoi(argv[2]);
    num_iterations = atoi(argv[3]);
    divergence = atoi(argv[4]);
    int stride = atoi(argv[5]);

    N = GLOBAL_MEM_ELEMENTS;
    parametric_measure_shared(N, 10, stride);

    return 0;
}
