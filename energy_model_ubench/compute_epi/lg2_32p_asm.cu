#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime_api.h>
//#include <cutil.h>
#include <hip/hip_runtime.h>

float* h_A;
float* h_B;
float* h_C;
float* h_res;
float* d_A;
float* d_B;
float* d_C;
float* d_res;

__global__
//void compute(const float* A, const float* B, const float* C, float* D, int n) {
void compute(float* D, int n, int div) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    float I1 = tid * 2.0;

    int thread_id = threadIdx.x % 32;

    if (thread_id < div) {
        __asm volatile (
                " .reg .f32 %r12;\n\t"
                " .reg .f32 %r13;\n\t"
                " .reg .f32 %r14;\n\t"
                " .reg .f32 %r15;\n\t"
                " .reg .f32 %r16;\n\t"
                " .reg .f32 %r17;\n\t"
                " .reg .f32 %r18;\n\t"
                " .reg .f32 %r19;\n\t"
                " .reg .f32 %r20;\n\t"
                " .reg .f32 %r21;\n\t"
                " .reg .f32 %r22;\n\t"
                " .reg .f32 %r23;\n\t"
                " .reg .f32 %r24;\n\t"
                " .reg .f32 %r25;\n\t"
                " .reg .f32 %r26;\n\t"
                " .reg .f32 %r27;\n\t"
                " .reg .f32 %r28;\n\t"
                "mov.f32 %r12, 4.4;\n\t"
                "mov.f32 %r13, %r12;\n\t"
                "mov.f32 %r14, 2.2;\n\t"
                "mov.f32 %r15, 3.3;\n\t"
                "mov.f32 %r16, 1.23;\n\t"
                "mov.f32 %r17, 2.42;\n\t"
                "mov.f32 %r18, 3.34;\n\t"
                "mov.f32 %r19, 5.62;\n\t"
                "mov.f32 %r20, 2.56;\n\t"
                "mov.f32 %r21, 1.56;\n\t"
                "mov.f32 %r22, 2.56;\n\t"
                "mov.f32 %r23, 5.56;\n\t"
                "mov.f32 %r24, 8.56;\n\t"
                "mov.f32 %r25, 3.56;\n\t"
                "mov.f32 %r26, 5.56;\n\t"
                "mov.f32 %r27, 6.56;\n\t"
                "mov.f32 %r28, 5.6;\n\t"

                );
        for (int k = 0; k < n; k++) {
            __asm volatile (
                    "lg2.approx.f32 %r13, %r13;\n\t" 
                    "lg2.approx.f32 %r14, %r14;\n\t" 
                    "lg2.approx.f32 %r15, %r15;\n\t" 
                    "lg2.approx.f32 %r16, %r16;\n\t" 
                    "lg2.approx.f32 %r17, %r17;\n\t" 
                    "lg2.approx.f32 %r18, %r18;\n\t" 
                    "lg2.approx.f32 %r19, %r19;\n\t" 
                    "lg2.approx.f32 %r20, %r20;\n\t" 
                    "lg2.approx.f32 %r21, %r21;\n\t" 
                    "lg2.approx.f32 %r22, %r22;\n\t" 
                    "lg2.approx.f32 %r23, %r23;\n\t" 
                    "lg2.approx.f32 %r24, %r24;\n\t" 
                    "lg2.approx.f32 %r25, %r25;\n\t" 
                    "lg2.approx.f32 %r26, %r26;\n\t" 
                    "lg2.approx.f32 %r27, %r27;\n\t" 
                    "lg2.approx.f32 %r28, %r28;\n\t" 
                    "lg2.approx.f32 %r13, %r13;\n\t" 
                    "lg2.approx.f32 %r14, %r14;\n\t" 
                    "lg2.approx.f32 %r15, %r15;\n\t" 
                    "lg2.approx.f32 %r16, %r16;\n\t" 
                    "lg2.approx.f32 %r17, %r17;\n\t" 
                    "lg2.approx.f32 %r18, %r18;\n\t" 
                    "lg2.approx.f32 %r19, %r19;\n\t" 
                    "lg2.approx.f32 %r20, %r20;\n\t" 
                    "lg2.approx.f32 %r21, %r21;\n\t" 
                    "lg2.approx.f32 %r22, %r22;\n\t" 
                    "lg2.approx.f32 %r23, %r23;\n\t" 
                    "lg2.approx.f32 %r24, %r24;\n\t" 
                    "lg2.approx.f32 %r25, %r25;\n\t" 
                    "lg2.approx.f32 %r26, %r26;\n\t" 
                    "lg2.approx.f32 %r27, %r27;\n\t" 
                    "lg2.approx.f32 %r28, %r28;\n\t" 
                    "lg2.approx.f32 %r13, %r13;\n\t" 
                    "lg2.approx.f32 %r14, %r14;\n\t" 
                    "lg2.approx.f32 %r15, %r15;\n\t" 
                    "lg2.approx.f32 %r16, %r16;\n\t" 
                    "lg2.approx.f32 %r17, %r17;\n\t" 
                    "lg2.approx.f32 %r18, %r18;\n\t" 
                    "lg2.approx.f32 %r19, %r19;\n\t" 
                    "lg2.approx.f32 %r20, %r20;\n\t" 
                    "lg2.approx.f32 %r21, %r21;\n\t" 
                    "lg2.approx.f32 %r22, %r22;\n\t" 
                    "lg2.approx.f32 %r23, %r23;\n\t" 
                    "lg2.approx.f32 %r24, %r24;\n\t" 
                    "lg2.approx.f32 %r25, %r25;\n\t" 
                    "lg2.approx.f32 %r26, %r26;\n\t" 
                    "lg2.approx.f32 %r27, %r27;\n\t" 
                    "lg2.approx.f32 %r28, %r28;\n\t" 
                    "lg2.approx.f32 %r13, %r13;\n\t" 
                    "lg2.approx.f32 %r14, %r14;\n\t" 
                    "lg2.approx.f32 %r15, %r15;\n\t" 
                    "lg2.approx.f32 %r16, %r16;\n\t" 
                    "lg2.approx.f32 %r17, %r17;\n\t" 
                    "lg2.approx.f32 %r18, %r18;\n\t" 
                    "lg2.approx.f32 %r19, %r19;\n\t" 
                    "lg2.approx.f32 %r20, %r20;\n\t" 
                    "lg2.approx.f32 %r21, %r21;\n\t" 
                    "lg2.approx.f32 %r22, %r22;\n\t" 
                    "lg2.approx.f32 %r23, %r23;\n\t" 
                    "lg2.approx.f32 %r24, %r24;\n\t" 
                    "lg2.approx.f32 %r25, %r25;\n\t" 
                    "lg2.approx.f32 %r26, %r26;\n\t" 
                    "lg2.approx.f32 %r27, %r27;\n\t" 
                    "lg2.approx.f32 %r28, %r28;\n\t" 
                    "lg2.approx.f32 %r13, %r13;\n\t" 
                    "lg2.approx.f32 %r14, %r14;\n\t" 
                    "lg2.approx.f32 %r15, %r15;\n\t" 
                    "lg2.approx.f32 %r16, %r16;\n\t" 
                    "lg2.approx.f32 %r17, %r17;\n\t" 
                    "lg2.approx.f32 %r18, %r18;\n\t" 
                    "lg2.approx.f32 %r19, %r19;\n\t" 
                    "lg2.approx.f32 %r20, %r20;\n\t" 
                    "lg2.approx.f32 %r21, %r21;\n\t" 
                    "lg2.approx.f32 %r22, %r22;\n\t" 
                    "lg2.approx.f32 %r23, %r23;\n\t" 
                    "lg2.approx.f32 %r24, %r24;\n\t" 
                    "lg2.approx.f32 %r25, %r25;\n\t" 
                    "lg2.approx.f32 %r26, %r26;\n\t" 
                    "lg2.approx.f32 %r27, %r27;\n\t" 
                    "lg2.approx.f32 %r28, %r28;\n\t" 
                    "lg2.approx.f32 %r13, %r13;\n\t" 
                    "lg2.approx.f32 %r14, %r14;\n\t" 
                    "lg2.approx.f32 %r15, %r15;\n\t" 
                    "lg2.approx.f32 %r16, %r16;\n\t" 
                    "lg2.approx.f32 %r17, %r17;\n\t" 
                    "lg2.approx.f32 %r18, %r18;\n\t" 
                    "lg2.approx.f32 %r19, %r19;\n\t" 
                    "lg2.approx.f32 %r20, %r20;\n\t" 
                    "lg2.approx.f32 %r21, %r21;\n\t" 
                    "lg2.approx.f32 %r22, %r22;\n\t" 
                    "lg2.approx.f32 %r23, %r23;\n\t" 
                    "lg2.approx.f32 %r24, %r24;\n\t" 
                    "lg2.approx.f32 %r25, %r25;\n\t" 
                    "lg2.approx.f32 %r26, %r26;\n\t" 
                    "lg2.approx.f32 %r27, %r27;\n\t" 
                    "lg2.approx.f32 %r28, %r28;\n\t" 
                    "lg2.approx.f32 %r13, %r13;\n\t" 
                    "lg2.approx.f32 %r14, %r14;\n\t" 
                    "lg2.approx.f32 %r15, %r15;\n\t" 
                    "lg2.approx.f32 %r16, %r16;\n\t" 
                    "lg2.approx.f32 %r17, %r17;\n\t" 
                    "lg2.approx.f32 %r18, %r18;\n\t" 
                    "lg2.approx.f32 %r19, %r19;\n\t" 
                    "lg2.approx.f32 %r20, %r20;\n\t" 
                    "lg2.approx.f32 %r21, %r21;\n\t" 
                    "lg2.approx.f32 %r22, %r22;\n\t" 
                    "lg2.approx.f32 %r23, %r23;\n\t" 
                    "lg2.approx.f32 %r24, %r24;\n\t" 
                    "lg2.approx.f32 %r25, %r25;\n\t" 
                    "lg2.approx.f32 %r26, %r26;\n\t" 
                    "lg2.approx.f32 %r27, %r27;\n\t" 
                    "lg2.approx.f32 %r28, %r28;\n\t" 
                    "lg2.approx.f32 %r13, %r13;\n\t" 
                    "lg2.approx.f32 %r14, %r14;\n\t" 
                    "lg2.approx.f32 %r15, %r15;\n\t" 
                    "lg2.approx.f32 %r16, %r16;\n\t" 
                    "lg2.approx.f32 %r17, %r17;\n\t" 
                    "lg2.approx.f32 %r18, %r18;\n\t" 
                    "lg2.approx.f32 %r19, %r19;\n\t" 
                    "lg2.approx.f32 %r20, %r20;\n\t" 
                    "lg2.approx.f32 %r21, %r21;\n\t" 
                    "lg2.approx.f32 %r22, %r22;\n\t" 
                    "lg2.approx.f32 %r23, %r23;\n\t" 
                    "lg2.approx.f32 %r24, %r24;\n\t" 
                    "lg2.approx.f32 %r25, %r25;\n\t" 
                    "lg2.approx.f32 %r26, %r26;\n\t" 
                    "lg2.approx.f32 %r27, %r27;\n\t" 
                    "lg2.approx.f32 %r28, %r28;\n\t" 
                    "lg2.approx.f32 %r13, %r13;\n\t" 
                    "lg2.approx.f32 %r14, %r14;\n\t" 
                    "lg2.approx.f32 %r15, %r15;\n\t" 
                    "lg2.approx.f32 %r16, %r16;\n\t" 
                    "lg2.approx.f32 %r17, %r17;\n\t" 
                    "lg2.approx.f32 %r18, %r18;\n\t" 
                    "lg2.approx.f32 %r19, %r19;\n\t" 
                    "lg2.approx.f32 %r20, %r20;\n\t" 
                    "lg2.approx.f32 %r21, %r21;\n\t" 
                    "lg2.approx.f32 %r22, %r22;\n\t" 
                    "lg2.approx.f32 %r23, %r23;\n\t" 
                    "lg2.approx.f32 %r24, %r24;\n\t" 
                    "lg2.approx.f32 %r25, %r25;\n\t" 
                    "lg2.approx.f32 %r26, %r26;\n\t" 
                    "lg2.approx.f32 %r27, %r27;\n\t" 
                    "lg2.approx.f32 %r28, %r28;\n\t" 
                    "lg2.approx.f32 %r13, %r13;\n\t" 
                    "lg2.approx.f32 %r14, %r14;\n\t" 
                    "lg2.approx.f32 %r15, %r15;\n\t" 
                    "lg2.approx.f32 %r16, %r16;\n\t" 
                    "lg2.approx.f32 %r17, %r17;\n\t" 
                    "lg2.approx.f32 %r18, %r18;\n\t" 
                    "lg2.approx.f32 %r19, %r19;\n\t" 
                    "lg2.approx.f32 %r20, %r20;\n\t" 
                    "lg2.approx.f32 %r21, %r21;\n\t" 
                    "lg2.approx.f32 %r22, %r22;\n\t" 
                    "lg2.approx.f32 %r23, %r23;\n\t" 
                    "lg2.approx.f32 %r24, %r24;\n\t" 
                    "lg2.approx.f32 %r25, %r25;\n\t" 
                    "lg2.approx.f32 %r26, %r26;\n\t" 
                    "lg2.approx.f32 %r27, %r27;\n\t" 
                    "lg2.approx.f32 %r28, %r28;\n\t" 
                    "lg2.approx.f32 %r13, %r13;\n\t" 
                    "lg2.approx.f32 %r14, %r14;\n\t" 
                    "lg2.approx.f32 %r15, %r15;\n\t" 
                    "lg2.approx.f32 %r16, %r16;\n\t" 
                    "lg2.approx.f32 %r17, %r17;\n\t" 
                    "lg2.approx.f32 %r18, %r18;\n\t" 
                    "lg2.approx.f32 %r19, %r19;\n\t" 
                    "lg2.approx.f32 %r20, %r20;\n\t" 
                    "lg2.approx.f32 %r21, %r21;\n\t" 
                    "lg2.approx.f32 %r22, %r22;\n\t" 
                    "lg2.approx.f32 %r23, %r23;\n\t" 
                    "lg2.approx.f32 %r24, %r24;\n\t" 
                    "lg2.approx.f32 %r25, %r25;\n\t" 
                    "lg2.approx.f32 %r26, %r26;\n\t" 
                    "lg2.approx.f32 %r27, %r27;\n\t" 
                    "lg2.approx.f32 %r28, %r28;\n\t" 
                    "lg2.approx.f32 %r13, %r13;\n\t" 
                    "lg2.approx.f32 %r14, %r14;\n\t" 
                    "lg2.approx.f32 %r15, %r15;\n\t" 
                    "lg2.approx.f32 %r16, %r16;\n\t" 
                    "lg2.approx.f32 %r17, %r17;\n\t" 
                    "lg2.approx.f32 %r18, %r18;\n\t" 
                    "lg2.approx.f32 %r19, %r19;\n\t" 
                    "lg2.approx.f32 %r20, %r20;\n\t" 
                    "lg2.approx.f32 %r21, %r21;\n\t" 
                    "lg2.approx.f32 %r22, %r22;\n\t" 
                    "lg2.approx.f32 %r23, %r23;\n\t" 
                    "lg2.approx.f32 %r24, %r24;\n\t" 
                    "lg2.approx.f32 %r25, %r25;\n\t" 
                    "lg2.approx.f32 %r26, %r26;\n\t" 
                    "lg2.approx.f32 %r27, %r27;\n\t" 
                    "lg2.approx.f32 %r28, %r28;\n\t" 
                    "lg2.approx.f32 %r13, %r13;\n\t" 
                    "lg2.approx.f32 %r14, %r14;\n\t" 
                    "lg2.approx.f32 %r15, %r15;\n\t" 
                    "lg2.approx.f32 %r16, %r16;\n\t" 
                    "lg2.approx.f32 %r17, %r17;\n\t" 
                    "lg2.approx.f32 %r18, %r18;\n\t" 
                    "lg2.approx.f32 %r19, %r19;\n\t" 
                    "lg2.approx.f32 %r20, %r20;\n\t" 
                    "lg2.approx.f32 %r21, %r21;\n\t" 
                    "lg2.approx.f32 %r22, %r22;\n\t" 
                    "lg2.approx.f32 %r23, %r23;\n\t" 
                    "lg2.approx.f32 %r24, %r24;\n\t" 
                    "lg2.approx.f32 %r25, %r25;\n\t" 
                    "lg2.approx.f32 %r26, %r26;\n\t" 
                    "lg2.approx.f32 %r27, %r27;\n\t" 
                    "lg2.approx.f32 %r28, %r28;\n\t" 
                    "lg2.approx.f32 %r13, %r13;\n\t" 
                    "lg2.approx.f32 %r14, %r14;\n\t" 
                    "lg2.approx.f32 %r15, %r15;\n\t" 
                    "lg2.approx.f32 %r16, %r16;\n\t" 
                    "lg2.approx.f32 %r17, %r17;\n\t" 
                    "lg2.approx.f32 %r18, %r18;\n\t" 
                    "lg2.approx.f32 %r19, %r19;\n\t" 
                    "lg2.approx.f32 %r20, %r20;\n\t" 
                    "lg2.approx.f32 %r21, %r21;\n\t" 
                    "lg2.approx.f32 %r22, %r22;\n\t" 
                    "lg2.approx.f32 %r23, %r23;\n\t" 
                    "lg2.approx.f32 %r24, %r24;\n\t" 
                    "lg2.approx.f32 %r25, %r25;\n\t" 
                    "lg2.approx.f32 %r26, %r26;\n\t" 
                    "lg2.approx.f32 %r27, %r27;\n\t" 
                    "lg2.approx.f32 %r28, %r28;\n\t" 
                    "lg2.approx.f32 %r13, %r13;\n\t" 
                    "lg2.approx.f32 %r14, %r14;\n\t" 
                    "lg2.approx.f32 %r15, %r15;\n\t" 
                    "lg2.approx.f32 %r16, %r16;\n\t" 
                    "lg2.approx.f32 %r17, %r17;\n\t" 
                    "lg2.approx.f32 %r18, %r18;\n\t" 
                    "lg2.approx.f32 %r19, %r19;\n\t" 
                    "lg2.approx.f32 %r20, %r20;\n\t" 
                    "lg2.approx.f32 %r21, %r21;\n\t" 
                    "lg2.approx.f32 %r22, %r22;\n\t" 
                    "lg2.approx.f32 %r23, %r23;\n\t" 
                    "lg2.approx.f32 %r24, %r24;\n\t" 
                    "lg2.approx.f32 %r25, %r25;\n\t" 
                    "lg2.approx.f32 %r26, %r26;\n\t" 
                    "lg2.approx.f32 %r27, %r27;\n\t" 
                    "lg2.approx.f32 %r28, %r28;\n\t" 
                    "lg2.approx.f32 %r13, %r13;\n\t" 
                    "lg2.approx.f32 %r14, %r14;\n\t" 
                    "lg2.approx.f32 %r15, %r15;\n\t" 
                    "lg2.approx.f32 %r16, %r16;\n\t" 
                    "lg2.approx.f32 %r17, %r17;\n\t" 
                    "lg2.approx.f32 %r18, %r18;\n\t" 
                    "lg2.approx.f32 %r19, %r19;\n\t" 
                    "lg2.approx.f32 %r20, %r20;\n\t" 
                    "lg2.approx.f32 %r21, %r21;\n\t" 
                    "lg2.approx.f32 %r22, %r22;\n\t" 
                    "lg2.approx.f32 %r23, %r23;\n\t" 
                    "lg2.approx.f32 %r24, %r24;\n\t" 
                    "lg2.approx.f32 %r25, %r25;\n\t" 
                    "lg2.approx.f32 %r26, %r26;\n\t" 
                    "lg2.approx.f32 %r27, %r27;\n\t" 
                    "lg2.approx.f32 %r28, %r28;\n\t" 
                    );
        }
    }
//    __syncthreads();

    //    if ((blockDim.x * blockIdx.x + threadIdx.x) == 0)
    *D = I1;

//    __syncthreads();
}

void usage() {
    std::cout << "Usage ./binary <num_blocks> <num_threads_per_block> <iterations>" "threads active per warp" << std::endl;
}

int main(int argc, char **argv)
{
    if (argc != 5) {
        usage();
        exit(1);
    }

    int num_blocks = atoi(argv[1]);
    int num_threads_per_block = atoi(argv[2]);
    int iterations = atoi(argv[3]);
    int divergence = atoi(argv[4]);

//    h_A = new float(2.0);
//    h_B = new float(3.0);
//    h_C = new float(4.0);

//    cudaMalloc((void**)&d_A, sizeof(float));
//    cudaMalloc((void**)&d_B, sizeof(float));
//    cudaMalloc((void**)&d_C, sizeof(float));
    hipMalloc((void**)&d_res, sizeof(float));
  
//    cudaMemcpy(d_A, h_A, sizeof(float), cudaMemcpyHostToDevice);
//    cudaMemcpy(d_B, h_B, sizeof(float), cudaMemcpyHostToDevice);
//    cudaMemcpy(d_C, h_C, sizeof(float), cudaMemcpyHostToDevice);
     
    hipEvent_t start, stop;
    float time;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    hipProfilerStart();
    
//    compute<<<num_blocks, num_threads_per_block>>>(d_A, d_B, d_C, d_res, iterations);
    compute<<<num_blocks, num_threads_per_block>>>(d_res, iterations, divergence);

    hipProfilerStop();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&time, start, stop);

    std::cout << "GPU Elapsed Time = " << time << std::endl;
  
    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipDeviceSynchronize();

    hipMemcpy(h_res, d_res, sizeof(float), hipMemcpyDeviceToHost);

    return 0;
}
