#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime_api.h>
//#include <cutil.h>
#include <hip/hip_runtime.h>

float* h_A;
float* h_B;
float* h_C;
float* h_res;
float* d_A;
float* d_B;
float* d_C;
float* d_res;

__global__
//void compute(const float* A, const float* B, const float* C, float* D, int n) {
void compute(float* D, int n, int div) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    float I1 = tid * 2.0;

    int thread_id = threadIdx.x % 32;

    if (thread_id < div) {
        __asm volatile (
                " .reg .s32 %r12;\n\t"
                " .reg .s32 %r13;\n\t"
                " .reg .s32 %r14;\n\t"
                " .reg .s32 %r15;\n\t"
                " .reg .s32 %r16;\n\t"
                " .reg .s32 %r17;\n\t"
                " .reg .s32 %r18;\n\t"
                " .reg .s32 %r19;\n\t"
                " .reg .s32 %r20;\n\t"
                " .reg .s32 %r21;\n\t"
                " .reg .s32 %r22;\n\t"
                " .reg .s32 %r23;\n\t"
                " .reg .s32 %r24;\n\t"
                " .reg .s32 %r25;\n\t"
                " .reg .s32 %r26;\n\t"
                " .reg .s32 %r27;\n\t"
                " .reg .s32 %r28;\n\t"
                "mov.s32 %r12, 44;\n\t"
                "mov.s32 %r13, %r12;\n\t"
                "mov.s32 %r14, 22;\n\t"
                "mov.s32 %r15, 33;\n\t"
                "mov.s32 %r16, 123;\n\t"
                "mov.s32 %r17, 242;\n\t"
                "mov.s32 %r18, 334;\n\t"
                "mov.s32 %r19, 562;\n\t"
                "mov.s32 %r20, 256;\n\t"
                "mov.s32 %r21, 156;\n\t"
                "mov.s32 %r22, 256;\n\t"
                "mov.s32 %r23, 556;\n\t"
                "mov.s32 %r24, 856;\n\t"
                "mov.s32 %r25, 356;\n\t"
                "mov.s32 %r26, 556;\n\t"
                "mov.s32 %r27, 656;\n\t"
                "mov.s32 %r28, 56;\n\t"

                );
        for (int k = 0; k < n; k++) {
            __asm volatile (
                    "add.s32 %r13, %r11, %r13;\n\t" 
                    "add.s32 %r14, %r11, %r14;\n\t" 
                    "add.s32 %r15, %r11, %r15;\n\t" 
                    "add.s32 %r16, %r11, %r16;\n\t" 
                    "add.s32 %r17, %r11, %r17;\n\t" 
                    "add.s32 %r18, %r11, %r18;\n\t" 
                    "add.s32 %r19, %r11, %r19;\n\t" 
                    "add.s32 %r20, %r11, %r20;\n\t" 
                    "add.s32 %r21, %r11, %r21;\n\t" 
                    "add.s32 %r22, %r11, %r22;\n\t" 
                    "add.s32 %r23, %r11, %r23;\n\t" 
                    "add.s32 %r24, %r11, %r24;\n\t" 
                    "add.s32 %r25, %r11, %r25;\n\t" 
                    "add.s32 %r26, %r11, %r26;\n\t" 
                    "add.s32 %r27, %r11, %r27;\n\t" 
                    "add.s32 %r28, %r11, %r28;\n\t" 
                    "add.s32 %r13, %r11, %r13;\n\t" 
                    "add.s32 %r14, %r11, %r14;\n\t" 
                    "add.s32 %r15, %r11, %r15;\n\t" 
                    "add.s32 %r16, %r11, %r16;\n\t" 
                    "add.s32 %r17, %r11, %r17;\n\t" 
                    "add.s32 %r18, %r11, %r18;\n\t" 
                    "add.s32 %r19, %r11, %r19;\n\t" 
                    "add.s32 %r20, %r11, %r20;\n\t" 
                    "add.s32 %r21, %r11, %r21;\n\t" 
                    "add.s32 %r22, %r11, %r22;\n\t" 
                    "add.s32 %r23, %r11, %r23;\n\t" 
                    "add.s32 %r24, %r11, %r24;\n\t" 
                    "add.s32 %r25, %r11, %r25;\n\t" 
                    "add.s32 %r26, %r11, %r26;\n\t" 
                    "add.s32 %r27, %r11, %r27;\n\t" 
                    "add.s32 %r28, %r11, %r28;\n\t" 
                    "add.s32 %r13, %r11, %r13;\n\t" 
                    "add.s32 %r14, %r11, %r14;\n\t" 
                    "add.s32 %r15, %r11, %r15;\n\t" 
                    "add.s32 %r16, %r11, %r16;\n\t" 
                    "add.s32 %r17, %r11, %r17;\n\t" 
                    "add.s32 %r18, %r11, %r18;\n\t" 
                    "add.s32 %r19, %r11, %r19;\n\t" 
                    "add.s32 %r20, %r11, %r20;\n\t" 
                    "add.s32 %r21, %r11, %r21;\n\t" 
                    "add.s32 %r22, %r11, %r22;\n\t" 
                    "add.s32 %r23, %r11, %r23;\n\t" 
                    "add.s32 %r24, %r11, %r24;\n\t" 
                    "add.s32 %r25, %r11, %r25;\n\t" 
                    "add.s32 %r26, %r11, %r26;\n\t" 
                    "add.s32 %r27, %r11, %r27;\n\t" 
                    "add.s32 %r28, %r11, %r28;\n\t" 
                    "add.s32 %r13, %r11, %r13;\n\t" 
                    "add.s32 %r14, %r11, %r14;\n\t" 
                    "add.s32 %r15, %r11, %r15;\n\t" 
                    "add.s32 %r16, %r11, %r16;\n\t" 
                    "add.s32 %r17, %r11, %r17;\n\t" 
                    "add.s32 %r18, %r11, %r18;\n\t" 
                    "add.s32 %r19, %r11, %r19;\n\t" 
                    "add.s32 %r20, %r11, %r20;\n\t" 
                    "add.s32 %r21, %r11, %r21;\n\t" 
                    "add.s32 %r22, %r11, %r22;\n\t" 
                    "add.s32 %r23, %r11, %r23;\n\t" 
                    "add.s32 %r24, %r11, %r24;\n\t" 
                    "add.s32 %r25, %r11, %r25;\n\t" 
                    "add.s32 %r26, %r11, %r26;\n\t" 
                    "add.s32 %r27, %r11, %r27;\n\t" 
                    "add.s32 %r28, %r11, %r28;\n\t" 
                    "add.s32 %r13, %r11, %r13;\n\t" 
                    "add.s32 %r14, %r11, %r14;\n\t" 
                    "add.s32 %r15, %r11, %r15;\n\t" 
                    "add.s32 %r16, %r11, %r16;\n\t" 
                    "add.s32 %r17, %r11, %r17;\n\t" 
                    "add.s32 %r18, %r11, %r18;\n\t" 
                    "add.s32 %r19, %r11, %r19;\n\t" 
                    "add.s32 %r20, %r11, %r20;\n\t" 
                    "add.s32 %r21, %r11, %r21;\n\t" 
                    "add.s32 %r22, %r11, %r22;\n\t" 
                    "add.s32 %r23, %r11, %r23;\n\t" 
                    "add.s32 %r24, %r11, %r24;\n\t" 
                    "add.s32 %r25, %r11, %r25;\n\t" 
                    "add.s32 %r26, %r11, %r26;\n\t" 
                    "add.s32 %r27, %r11, %r27;\n\t" 
                    "add.s32 %r28, %r11, %r28;\n\t" 
                    "add.s32 %r13, %r11, %r13;\n\t" 
                    "add.s32 %r14, %r11, %r14;\n\t" 
                    "add.s32 %r15, %r11, %r15;\n\t" 
                    "add.s32 %r16, %r11, %r16;\n\t" 
                    "add.s32 %r17, %r11, %r17;\n\t" 
                    "add.s32 %r18, %r11, %r18;\n\t" 
                    "add.s32 %r19, %r11, %r19;\n\t" 
                    "add.s32 %r20, %r11, %r20;\n\t" 
                    "add.s32 %r21, %r11, %r21;\n\t" 
                    "add.s32 %r22, %r11, %r22;\n\t" 
                    "add.s32 %r23, %r11, %r23;\n\t" 
                    "add.s32 %r24, %r11, %r24;\n\t" 
                    "add.s32 %r25, %r11, %r25;\n\t" 
                    "add.s32 %r26, %r11, %r26;\n\t" 
                    "add.s32 %r27, %r11, %r27;\n\t" 
                    "add.s32 %r28, %r11, %r28;\n\t" 
                    "add.s32 %r13, %r11, %r13;\n\t" 
                    "add.s32 %r14, %r11, %r14;\n\t" 
                    "add.s32 %r15, %r11, %r15;\n\t" 
                    "add.s32 %r16, %r11, %r16;\n\t" 
                    "add.s32 %r17, %r11, %r17;\n\t" 
                    "add.s32 %r18, %r11, %r18;\n\t" 
                    "add.s32 %r19, %r11, %r19;\n\t" 
                    "add.s32 %r20, %r11, %r20;\n\t" 
                    "add.s32 %r21, %r11, %r21;\n\t" 
                    "add.s32 %r22, %r11, %r22;\n\t" 
                    "add.s32 %r23, %r11, %r23;\n\t" 
                    "add.s32 %r24, %r11, %r24;\n\t" 
                    "add.s32 %r25, %r11, %r25;\n\t" 
                    "add.s32 %r26, %r11, %r26;\n\t" 
                    "add.s32 %r27, %r11, %r27;\n\t" 
                    "add.s32 %r28, %r11, %r28;\n\t" 
                    "add.s32 %r13, %r11, %r13;\n\t" 
                    "add.s32 %r14, %r11, %r14;\n\t" 
                    "add.s32 %r15, %r11, %r15;\n\t" 
                    "add.s32 %r16, %r11, %r16;\n\t" 
                    "add.s32 %r17, %r11, %r17;\n\t" 
                    "add.s32 %r18, %r11, %r18;\n\t" 
                    "add.s32 %r19, %r11, %r19;\n\t" 
                    "add.s32 %r20, %r11, %r20;\n\t" 
                    "add.s32 %r21, %r11, %r21;\n\t" 
                    "add.s32 %r22, %r11, %r22;\n\t" 
                    "add.s32 %r23, %r11, %r23;\n\t" 
                    "add.s32 %r24, %r11, %r24;\n\t" 
                    "add.s32 %r25, %r11, %r25;\n\t" 
                    "add.s32 %r26, %r11, %r26;\n\t" 
                    "add.s32 %r27, %r11, %r27;\n\t" 
                    "add.s32 %r28, %r11, %r28;\n\t" 
                    "add.s32 %r13, %r11, %r13;\n\t" 
                    "add.s32 %r14, %r11, %r14;\n\t" 
                    "add.s32 %r15, %r11, %r15;\n\t" 
                    "add.s32 %r16, %r11, %r16;\n\t" 
                    "add.s32 %r17, %r11, %r17;\n\t" 
                    "add.s32 %r18, %r11, %r18;\n\t" 
                    "add.s32 %r19, %r11, %r19;\n\t" 
                    "add.s32 %r20, %r11, %r20;\n\t" 
                    "add.s32 %r21, %r11, %r21;\n\t" 
                    "add.s32 %r22, %r11, %r22;\n\t" 
                    "add.s32 %r23, %r11, %r23;\n\t" 
                    "add.s32 %r24, %r11, %r24;\n\t" 
                    "add.s32 %r25, %r11, %r25;\n\t" 
                    "add.s32 %r26, %r11, %r26;\n\t" 
                    "add.s32 %r27, %r11, %r27;\n\t" 
                    "add.s32 %r28, %r11, %r28;\n\t" 
                    "add.s32 %r13, %r11, %r13;\n\t" 
                    "add.s32 %r14, %r11, %r14;\n\t" 
                    "add.s32 %r15, %r11, %r15;\n\t" 
                    "add.s32 %r16, %r11, %r16;\n\t" 
                    "add.s32 %r17, %r11, %r17;\n\t" 
                    "add.s32 %r18, %r11, %r18;\n\t" 
                    "add.s32 %r19, %r11, %r19;\n\t" 
                    "add.s32 %r20, %r11, %r20;\n\t" 
                    "add.s32 %r21, %r11, %r21;\n\t" 
                    "add.s32 %r22, %r11, %r22;\n\t" 
                    "add.s32 %r23, %r11, %r23;\n\t" 
                    "add.s32 %r24, %r11, %r24;\n\t" 
                    "add.s32 %r25, %r11, %r25;\n\t" 
                    "add.s32 %r26, %r11, %r26;\n\t" 
                    "add.s32 %r27, %r11, %r27;\n\t" 
                    "add.s32 %r28, %r11, %r28;\n\t" 
                    "add.s32 %r13, %r11, %r13;\n\t" 
                    "add.s32 %r14, %r11, %r14;\n\t" 
                    "add.s32 %r15, %r11, %r15;\n\t" 
                    "add.s32 %r16, %r11, %r16;\n\t" 
                    "add.s32 %r17, %r11, %r17;\n\t" 
                    "add.s32 %r18, %r11, %r18;\n\t" 
                    "add.s32 %r19, %r11, %r19;\n\t" 
                    "add.s32 %r20, %r11, %r20;\n\t" 
                    "add.s32 %r21, %r11, %r21;\n\t" 
                    "add.s32 %r22, %r11, %r22;\n\t" 
                    "add.s32 %r23, %r11, %r23;\n\t" 
                    "add.s32 %r24, %r11, %r24;\n\t" 
                    "add.s32 %r25, %r11, %r25;\n\t" 
                    "add.s32 %r26, %r11, %r26;\n\t" 
                    "add.s32 %r27, %r11, %r27;\n\t" 
                    "add.s32 %r28, %r11, %r28;\n\t" 
                    "add.s32 %r13, %r11, %r13;\n\t" 
                    "add.s32 %r14, %r11, %r14;\n\t" 
                    "add.s32 %r15, %r11, %r15;\n\t" 
                    "add.s32 %r16, %r11, %r16;\n\t" 
                    "add.s32 %r17, %r11, %r17;\n\t" 
                    "add.s32 %r18, %r11, %r18;\n\t" 
                    "add.s32 %r19, %r11, %r19;\n\t" 
                    "add.s32 %r20, %r11, %r20;\n\t" 
                    "add.s32 %r21, %r11, %r21;\n\t" 
                    "add.s32 %r22, %r11, %r22;\n\t" 
                    "add.s32 %r23, %r11, %r23;\n\t" 
                    "add.s32 %r24, %r11, %r24;\n\t" 
                    "add.s32 %r25, %r11, %r25;\n\t" 
                    "add.s32 %r26, %r11, %r26;\n\t" 
                    "add.s32 %r27, %r11, %r27;\n\t" 
                    "add.s32 %r28, %r11, %r28;\n\t" 
                    "add.s32 %r13, %r11, %r13;\n\t" 
                    "add.s32 %r14, %r11, %r14;\n\t" 
                    "add.s32 %r15, %r11, %r15;\n\t" 
                    "add.s32 %r16, %r11, %r16;\n\t" 
                    "add.s32 %r17, %r11, %r17;\n\t" 
                    "add.s32 %r18, %r11, %r18;\n\t" 
                    "add.s32 %r19, %r11, %r19;\n\t" 
                    "add.s32 %r20, %r11, %r20;\n\t" 
                    "add.s32 %r21, %r11, %r21;\n\t" 
                    "add.s32 %r22, %r11, %r22;\n\t" 
                    "add.s32 %r23, %r11, %r23;\n\t" 
                    "add.s32 %r24, %r11, %r24;\n\t" 
                    "add.s32 %r25, %r11, %r25;\n\t" 
                    "add.s32 %r26, %r11, %r26;\n\t" 
                    "add.s32 %r27, %r11, %r27;\n\t" 
                    "add.s32 %r28, %r11, %r28;\n\t" 
                    "add.s32 %r13, %r11, %r13;\n\t" 
                    "add.s32 %r14, %r11, %r14;\n\t" 
                    "add.s32 %r15, %r11, %r15;\n\t" 
                    "add.s32 %r16, %r11, %r16;\n\t" 
                    "add.s32 %r17, %r11, %r17;\n\t" 
                    "add.s32 %r18, %r11, %r18;\n\t" 
                    "add.s32 %r19, %r11, %r19;\n\t" 
                    "add.s32 %r20, %r11, %r20;\n\t" 
                    "add.s32 %r21, %r11, %r21;\n\t" 
                    "add.s32 %r22, %r11, %r22;\n\t" 
                    "add.s32 %r23, %r11, %r23;\n\t" 
                    "add.s32 %r24, %r11, %r24;\n\t" 
                    "add.s32 %r25, %r11, %r25;\n\t" 
                    "add.s32 %r26, %r11, %r26;\n\t" 
                    "add.s32 %r27, %r11, %r27;\n\t" 
                    "add.s32 %r28, %r11, %r28;\n\t" 
                    "add.s32 %r13, %r11, %r13;\n\t" 
                    "add.s32 %r14, %r11, %r14;\n\t" 
                    "add.s32 %r15, %r11, %r15;\n\t" 
                    "add.s32 %r16, %r11, %r16;\n\t" 
                    "add.s32 %r17, %r11, %r17;\n\t" 
                    "add.s32 %r18, %r11, %r18;\n\t" 
                    "add.s32 %r19, %r11, %r19;\n\t" 
                    "add.s32 %r20, %r11, %r20;\n\t" 
                    "add.s32 %r21, %r11, %r21;\n\t" 
                    "add.s32 %r22, %r11, %r22;\n\t" 
                    "add.s32 %r23, %r11, %r23;\n\t" 
                    "add.s32 %r24, %r11, %r24;\n\t" 
                    "add.s32 %r25, %r11, %r25;\n\t" 
                    "add.s32 %r26, %r11, %r26;\n\t" 
                    "add.s32 %r27, %r11, %r27;\n\t" 
                    "add.s32 %r28, %r11, %r28;\n\t" 
                    "add.s32 %r13, %r11, %r13;\n\t" 
                    "add.s32 %r14, %r11, %r14;\n\t" 
                    "add.s32 %r15, %r11, %r15;\n\t" 
                    "add.s32 %r16, %r11, %r16;\n\t" 
                    "add.s32 %r17, %r11, %r17;\n\t" 
                    "add.s32 %r18, %r11, %r18;\n\t" 
                    "add.s32 %r19, %r11, %r19;\n\t" 
                    "add.s32 %r20, %r11, %r20;\n\t" 
                    "add.s32 %r21, %r11, %r21;\n\t" 
                    "add.s32 %r22, %r11, %r22;\n\t" 
                    "add.s32 %r23, %r11, %r23;\n\t" 
                    "add.s32 %r24, %r11, %r24;\n\t" 
                    "add.s32 %r25, %r11, %r25;\n\t" 
                    "add.s32 %r26, %r11, %r26;\n\t" 
                    "add.s32 %r27, %r11, %r27;\n\t" 
                    "add.s32 %r28, %r11, %r28;\n\t" 
                    );
        }
    }
//    __syncthreads();

    //    if ((blockDim.x * blockIdx.x + threadIdx.x) == 0)
    *D = I1;

//    __syncthreads();
}

void usage() {
    std::cout << "Usage ./binary <num_blocks> <num_threads_per_block> <iterations>" "threads active per warp" << std::endl;
}

int main(int argc, char **argv)
{
    if (argc != 5) {
        usage();
        exit(1);
    }

    int num_blocks = atoi(argv[1]);
    int num_threads_per_block = atoi(argv[2]);
    int iterations = atoi(argv[3]);
    int divergence = atoi(argv[4]);

//    h_A = new float(2.0);
//    h_B = new float(3.0);
//    h_C = new float(4.0);

//    cudaMalloc((void**)&d_A, sizeof(float));
//    cudaMalloc((void**)&d_B, sizeof(float));
//    cudaMalloc((void**)&d_C, sizeof(float));
    hipMalloc((void**)&d_res, sizeof(float));
  
//    cudaMemcpy(d_A, h_A, sizeof(float), cudaMemcpyHostToDevice);
//    cudaMemcpy(d_B, h_B, sizeof(float), cudaMemcpyHostToDevice);
//    cudaMemcpy(d_C, h_C, sizeof(float), cudaMemcpyHostToDevice);
     
    hipEvent_t start, stop;
    float time;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    hipProfilerStart();
    
//    compute<<<num_blocks, num_threads_per_block>>>(d_A, d_B, d_C, d_res, iterations);
    compute<<<num_blocks, num_threads_per_block>>>(d_res, iterations, divergence);

    hipProfilerStop();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&time, start, stop);

    std::cout << "GPU Elapsed Time = " << time << std::endl;
  
    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipDeviceSynchronize();

    hipMemcpy(h_res, d_res, sizeof(float), hipMemcpyDeviceToHost);

    return 0;
}
