#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime_api.h>
//#include <cutil.h>
#include <hip/hip_runtime.h>

float* h_A;
float* h_B;
float* h_C;
float* h_res;
float* d_A;
float* d_B;
float* d_C;
float* d_res;

__global__
//void compute(const float* A, const float* B, const float* C, float* D, int n) {
void compute(float* D, int n, int div) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    float I1 = tid * 2.0;

    int thread_id = threadIdx.x % 32;

    if (thread_id < div) {
        __asm volatile (
                " .reg .f64 %r12;\n\t"
                " .reg .f64 %r13;\n\t"
                " .reg .f64 %r14;\n\t"
                " .reg .f64 %r15;\n\t"
                " .reg .f64 %r16;\n\t"
                " .reg .f64 %r17;\n\t"
                " .reg .f64 %r18;\n\t"
                " .reg .f64 %r19;\n\t"
                " .reg .f64 %r20;\n\t"
                " .reg .f64 %r21;\n\t"
                " .reg .f64 %r22;\n\t"
                " .reg .f64 %r23;\n\t"
                " .reg .f64 %r24;\n\t"
                " .reg .f64 %r25;\n\t"
                " .reg .f64 %r26;\n\t"
                " .reg .f64 %r27;\n\t"
                " .reg .f64 %r28;\n\t"
                "mov.f64 %r12, 4.4;\n\t"
                "mov.f64 %r13, %r12;\n\t"
                "mov.f64 %r14, 2.2;\n\t"
                "mov.f64 %r15, 3.3;\n\t"
                "mov.f64 %r16, 1.23;\n\t"
                "mov.f64 %r17, 2.42;\n\t"
                "mov.f64 %r18, 3.34;\n\t"
                "mov.f64 %r19, 5.62;\n\t"
                "mov.f64 %r20, 2.56;\n\t"
                "mov.f64 %r21, 1.56;\n\t"
                "mov.f64 %r22, 2.56;\n\t"
                "mov.f64 %r23, 5.56;\n\t"
                "mov.f64 %r24, 8.56;\n\t"
                "mov.f64 %r25, 3.56;\n\t"
                "mov.f64 %r26, 5.56;\n\t"
                "mov.f64 %r27, 6.56;\n\t"
                "mov.f64 %r28, 0.56;\n\t"

                );
        for (int k = 0; k < n; k++) {
            __asm volatile (
                    "mul.rn.f64 %r13, %r12, %r13;\n\t" 
                    "mul.rn.f64 %r14, %r12, %r14;\n\t" 
                    "mul.rn.f64 %r15, %r12, %r15;\n\t" 
                    "mul.rn.f64 %r16, %r12, %r16;\n\t" 
                    "mul.rn.f64 %r17, %r12, %r17;\n\t" 
                    "mul.rn.f64 %r18, %r12, %r18;\n\t" 
                    "mul.rn.f64 %r19, %r12, %r19;\n\t" 
                    "mul.rn.f64 %r20, %r12, %r20;\n\t" 
                    "mul.rn.f64 %r21, %r12, %r21;\n\t" 
                    "mul.rn.f64 %r22, %r12, %r22;\n\t" 
                    "mul.rn.f64 %r23, %r12, %r23;\n\t" 
                    "mul.rn.f64 %r24, %r12, %r24;\n\t" 
                    "mul.rn.f64 %r25, %r12, %r25;\n\t" 
                    "mul.rn.f64 %r26, %r12, %r26;\n\t" 
                    "mul.rn.f64 %r27, %r12, %r27;\n\t" 
                    "mul.rn.f64 %r28, %r12, %r28;\n\t" 
                    "mul.rn.f64 %r13, %r12, %r13;\n\t" 
                    "mul.rn.f64 %r14, %r12, %r14;\n\t" 
                    "mul.rn.f64 %r15, %r12, %r15;\n\t" 
                    "mul.rn.f64 %r16, %r12, %r16;\n\t" 
                    "mul.rn.f64 %r17, %r12, %r17;\n\t" 
                    "mul.rn.f64 %r18, %r12, %r18;\n\t" 
                    "mul.rn.f64 %r19, %r12, %r19;\n\t" 
                    "mul.rn.f64 %r20, %r12, %r20;\n\t" 
                    "mul.rn.f64 %r21, %r12, %r21;\n\t" 
                    "mul.rn.f64 %r22, %r12, %r22;\n\t" 
                    "mul.rn.f64 %r23, %r12, %r23;\n\t" 
                    "mul.rn.f64 %r24, %r12, %r24;\n\t" 
                    "mul.rn.f64 %r25, %r12, %r25;\n\t" 
                    "mul.rn.f64 %r26, %r12, %r26;\n\t" 
                    "mul.rn.f64 %r27, %r12, %r27;\n\t" 
                    "mul.rn.f64 %r28, %r12, %r28;\n\t" 
                    "mul.rn.f64 %r13, %r12, %r13;\n\t" 
                    "mul.rn.f64 %r14, %r12, %r14;\n\t" 
                    "mul.rn.f64 %r15, %r12, %r15;\n\t" 
                    "mul.rn.f64 %r16, %r12, %r16;\n\t" 
                    "mul.rn.f64 %r17, %r12, %r17;\n\t" 
                    "mul.rn.f64 %r18, %r12, %r18;\n\t" 
                    "mul.rn.f64 %r19, %r12, %r19;\n\t" 
                    "mul.rn.f64 %r20, %r12, %r20;\n\t" 
                    "mul.rn.f64 %r21, %r12, %r21;\n\t" 
                    "mul.rn.f64 %r22, %r12, %r22;\n\t" 
                    "mul.rn.f64 %r23, %r12, %r23;\n\t" 
                    "mul.rn.f64 %r24, %r12, %r24;\n\t" 
                    "mul.rn.f64 %r25, %r12, %r25;\n\t" 
                    "mul.rn.f64 %r26, %r12, %r26;\n\t" 
                    "mul.rn.f64 %r27, %r12, %r27;\n\t" 
                    "mul.rn.f64 %r28, %r12, %r28;\n\t" 
                    "mul.rn.f64 %r13, %r12, %r13;\n\t" 
                    "mul.rn.f64 %r14, %r12, %r14;\n\t" 
                    "mul.rn.f64 %r15, %r12, %r15;\n\t" 
                    "mul.rn.f64 %r16, %r12, %r16;\n\t" 
                    "mul.rn.f64 %r17, %r12, %r17;\n\t" 
                    "mul.rn.f64 %r18, %r12, %r18;\n\t" 
                    "mul.rn.f64 %r19, %r12, %r19;\n\t" 
                    "mul.rn.f64 %r20, %r12, %r20;\n\t" 
                    "mul.rn.f64 %r21, %r12, %r21;\n\t" 
                    "mul.rn.f64 %r22, %r12, %r22;\n\t" 
                    "mul.rn.f64 %r23, %r12, %r23;\n\t" 
                    "mul.rn.f64 %r24, %r12, %r24;\n\t" 
                    "mul.rn.f64 %r25, %r12, %r25;\n\t" 
                    "mul.rn.f64 %r26, %r12, %r26;\n\t" 
                    "mul.rn.f64 %r27, %r12, %r27;\n\t" 
                    "mul.rn.f64 %r28, %r12, %r28;\n\t" 
                    "mul.rn.f64 %r13, %r12, %r13;\n\t" 
                    "mul.rn.f64 %r14, %r12, %r14;\n\t" 
                    "mul.rn.f64 %r15, %r12, %r15;\n\t" 
                    "mul.rn.f64 %r16, %r12, %r16;\n\t" 
                    "mul.rn.f64 %r17, %r12, %r17;\n\t" 
                    "mul.rn.f64 %r18, %r12, %r18;\n\t" 
                    "mul.rn.f64 %r19, %r12, %r19;\n\t" 
                    "mul.rn.f64 %r20, %r12, %r20;\n\t" 
                    "mul.rn.f64 %r21, %r12, %r21;\n\t" 
                    "mul.rn.f64 %r22, %r12, %r22;\n\t" 
                    "mul.rn.f64 %r23, %r12, %r23;\n\t" 
                    "mul.rn.f64 %r24, %r12, %r24;\n\t" 
                    "mul.rn.f64 %r25, %r12, %r25;\n\t" 
                    "mul.rn.f64 %r26, %r12, %r26;\n\t" 
                    "mul.rn.f64 %r27, %r12, %r27;\n\t" 
                    "mul.rn.f64 %r28, %r12, %r28;\n\t" 
                    "mul.rn.f64 %r13, %r12, %r13;\n\t" 
                    "mul.rn.f64 %r14, %r12, %r14;\n\t" 
                    "mul.rn.f64 %r15, %r12, %r15;\n\t" 
                    "mul.rn.f64 %r16, %r12, %r16;\n\t" 
                    "mul.rn.f64 %r17, %r12, %r17;\n\t" 
                    "mul.rn.f64 %r18, %r12, %r18;\n\t" 
                    "mul.rn.f64 %r19, %r12, %r19;\n\t" 
                    "mul.rn.f64 %r20, %r12, %r20;\n\t" 
                    "mul.rn.f64 %r21, %r12, %r21;\n\t" 
                    "mul.rn.f64 %r22, %r12, %r22;\n\t" 
                    "mul.rn.f64 %r23, %r12, %r23;\n\t" 
                    "mul.rn.f64 %r24, %r12, %r24;\n\t" 
                    "mul.rn.f64 %r25, %r12, %r25;\n\t" 
                    "mul.rn.f64 %r26, %r12, %r26;\n\t" 
                    "mul.rn.f64 %r27, %r12, %r27;\n\t" 
                    "mul.rn.f64 %r28, %r12, %r28;\n\t" 
                    "mul.rn.f64 %r13, %r12, %r13;\n\t" 
                    "mul.rn.f64 %r14, %r12, %r14;\n\t" 
                    "mul.rn.f64 %r15, %r12, %r15;\n\t" 
                    "mul.rn.f64 %r16, %r12, %r16;\n\t" 
                    "mul.rn.f64 %r17, %r12, %r17;\n\t" 
                    "mul.rn.f64 %r18, %r12, %r18;\n\t" 
                    "mul.rn.f64 %r19, %r12, %r19;\n\t" 
                    "mul.rn.f64 %r20, %r12, %r20;\n\t" 
                    "mul.rn.f64 %r21, %r12, %r21;\n\t" 
                    "mul.rn.f64 %r22, %r12, %r22;\n\t" 
                    "mul.rn.f64 %r23, %r12, %r23;\n\t" 
                    "mul.rn.f64 %r24, %r12, %r24;\n\t" 
                    "mul.rn.f64 %r25, %r12, %r25;\n\t" 
                    "mul.rn.f64 %r26, %r12, %r26;\n\t" 
                    "mul.rn.f64 %r27, %r12, %r27;\n\t" 
                    "mul.rn.f64 %r28, %r12, %r28;\n\t" 
                    "mul.rn.f64 %r13, %r12, %r13;\n\t" 
                    "mul.rn.f64 %r14, %r12, %r14;\n\t" 
                    "mul.rn.f64 %r15, %r12, %r15;\n\t" 
                    "mul.rn.f64 %r16, %r12, %r16;\n\t" 
                    "mul.rn.f64 %r17, %r12, %r17;\n\t" 
                    "mul.rn.f64 %r18, %r12, %r18;\n\t" 
                    "mul.rn.f64 %r19, %r12, %r19;\n\t" 
                    "mul.rn.f64 %r20, %r12, %r20;\n\t" 
                    "mul.rn.f64 %r21, %r12, %r21;\n\t" 
                    "mul.rn.f64 %r22, %r12, %r22;\n\t" 
                    "mul.rn.f64 %r23, %r12, %r23;\n\t" 
                    "mul.rn.f64 %r24, %r12, %r24;\n\t" 
                    "mul.rn.f64 %r25, %r12, %r25;\n\t" 
                    "mul.rn.f64 %r26, %r12, %r26;\n\t" 
                    "mul.rn.f64 %r27, %r12, %r27;\n\t" 
                    "mul.rn.f64 %r28, %r12, %r28;\n\t" 
                    "mul.rn.f64 %r13, %r12, %r13;\n\t" 
                    "mul.rn.f64 %r14, %r12, %r14;\n\t" 
                    "mul.rn.f64 %r15, %r12, %r15;\n\t" 
                    "mul.rn.f64 %r16, %r12, %r16;\n\t" 
                    "mul.rn.f64 %r17, %r12, %r17;\n\t" 
                    "mul.rn.f64 %r18, %r12, %r18;\n\t" 
                    "mul.rn.f64 %r19, %r12, %r19;\n\t" 
                    "mul.rn.f64 %r20, %r12, %r20;\n\t" 
                    "mul.rn.f64 %r21, %r12, %r21;\n\t" 
                    "mul.rn.f64 %r22, %r12, %r22;\n\t" 
                    "mul.rn.f64 %r23, %r12, %r23;\n\t" 
                    "mul.rn.f64 %r24, %r12, %r24;\n\t" 
                    "mul.rn.f64 %r25, %r12, %r25;\n\t" 
                    "mul.rn.f64 %r26, %r12, %r26;\n\t" 
                    "mul.rn.f64 %r27, %r12, %r27;\n\t" 
                    "mul.rn.f64 %r28, %r12, %r28;\n\t" 
                    "mul.rn.f64 %r13, %r12, %r13;\n\t" 
                    "mul.rn.f64 %r14, %r12, %r14;\n\t" 
                    "mul.rn.f64 %r15, %r12, %r15;\n\t" 
                    "mul.rn.f64 %r16, %r12, %r16;\n\t" 
                    "mul.rn.f64 %r17, %r12, %r17;\n\t" 
                    "mul.rn.f64 %r18, %r12, %r18;\n\t" 
                    "mul.rn.f64 %r19, %r12, %r19;\n\t" 
                    "mul.rn.f64 %r20, %r12, %r20;\n\t" 
                    "mul.rn.f64 %r21, %r12, %r21;\n\t" 
                    "mul.rn.f64 %r22, %r12, %r22;\n\t" 
                    "mul.rn.f64 %r23, %r12, %r23;\n\t" 
                    "mul.rn.f64 %r24, %r12, %r24;\n\t" 
                    "mul.rn.f64 %r25, %r12, %r25;\n\t" 
                    "mul.rn.f64 %r26, %r12, %r26;\n\t" 
                    "mul.rn.f64 %r27, %r12, %r27;\n\t" 
                    "mul.rn.f64 %r28, %r12, %r28;\n\t" 
                    "mul.rn.f64 %r13, %r12, %r13;\n\t" 
                    "mul.rn.f64 %r14, %r12, %r14;\n\t" 
                    "mul.rn.f64 %r15, %r12, %r15;\n\t" 
                    "mul.rn.f64 %r16, %r12, %r16;\n\t" 
                    "mul.rn.f64 %r17, %r12, %r17;\n\t" 
                    "mul.rn.f64 %r18, %r12, %r18;\n\t" 
                    "mul.rn.f64 %r19, %r12, %r19;\n\t" 
                    "mul.rn.f64 %r20, %r12, %r20;\n\t" 
                    "mul.rn.f64 %r21, %r12, %r21;\n\t" 
                    "mul.rn.f64 %r22, %r12, %r22;\n\t" 
                    "mul.rn.f64 %r23, %r12, %r23;\n\t" 
                    "mul.rn.f64 %r24, %r12, %r24;\n\t" 
                    "mul.rn.f64 %r25, %r12, %r25;\n\t" 
                    "mul.rn.f64 %r26, %r12, %r26;\n\t" 
                    "mul.rn.f64 %r27, %r12, %r27;\n\t" 
                    "mul.rn.f64 %r28, %r12, %r28;\n\t" 
                    "mul.rn.f64 %r13, %r12, %r13;\n\t" 
                    "mul.rn.f64 %r14, %r12, %r14;\n\t" 
                    "mul.rn.f64 %r15, %r12, %r15;\n\t" 
                    "mul.rn.f64 %r16, %r12, %r16;\n\t" 
                    "mul.rn.f64 %r17, %r12, %r17;\n\t" 
                    "mul.rn.f64 %r18, %r12, %r18;\n\t" 
                    "mul.rn.f64 %r19, %r12, %r19;\n\t" 
                    "mul.rn.f64 %r20, %r12, %r20;\n\t" 
                    "mul.rn.f64 %r21, %r12, %r21;\n\t" 
                    "mul.rn.f64 %r22, %r12, %r22;\n\t" 
                    "mul.rn.f64 %r23, %r12, %r23;\n\t" 
                    "mul.rn.f64 %r24, %r12, %r24;\n\t" 
                    "mul.rn.f64 %r25, %r12, %r25;\n\t" 
                    "mul.rn.f64 %r26, %r12, %r26;\n\t" 
                    "mul.rn.f64 %r27, %r12, %r27;\n\t" 
                    "mul.rn.f64 %r28, %r12, %r28;\n\t" 
                    "mul.rn.f64 %r13, %r12, %r13;\n\t" 
                    "mul.rn.f64 %r14, %r12, %r14;\n\t" 
                    "mul.rn.f64 %r15, %r12, %r15;\n\t" 
                    "mul.rn.f64 %r16, %r12, %r16;\n\t" 
                    "mul.rn.f64 %r17, %r12, %r17;\n\t" 
                    "mul.rn.f64 %r18, %r12, %r18;\n\t" 
                    "mul.rn.f64 %r19, %r12, %r19;\n\t" 
                    "mul.rn.f64 %r20, %r12, %r20;\n\t" 
                    "mul.rn.f64 %r21, %r12, %r21;\n\t" 
                    "mul.rn.f64 %r22, %r12, %r22;\n\t" 
                    "mul.rn.f64 %r23, %r12, %r23;\n\t" 
                    "mul.rn.f64 %r24, %r12, %r24;\n\t" 
                    "mul.rn.f64 %r25, %r12, %r25;\n\t" 
                    "mul.rn.f64 %r26, %r12, %r26;\n\t" 
                    "mul.rn.f64 %r27, %r12, %r27;\n\t" 
                    "mul.rn.f64 %r28, %r12, %r28;\n\t" 
                    "mul.rn.f64 %r13, %r12, %r13;\n\t" 
                    "mul.rn.f64 %r14, %r12, %r14;\n\t" 
                    "mul.rn.f64 %r15, %r12, %r15;\n\t" 
                    "mul.rn.f64 %r16, %r12, %r16;\n\t" 
                    "mul.rn.f64 %r17, %r12, %r17;\n\t" 
                    "mul.rn.f64 %r18, %r12, %r18;\n\t" 
                    "mul.rn.f64 %r19, %r12, %r19;\n\t" 
                    "mul.rn.f64 %r20, %r12, %r20;\n\t" 
                    "mul.rn.f64 %r21, %r12, %r21;\n\t" 
                    "mul.rn.f64 %r22, %r12, %r22;\n\t" 
                    "mul.rn.f64 %r23, %r12, %r23;\n\t" 
                    "mul.rn.f64 %r24, %r12, %r24;\n\t" 
                    "mul.rn.f64 %r25, %r12, %r25;\n\t" 
                    "mul.rn.f64 %r26, %r12, %r26;\n\t" 
                    "mul.rn.f64 %r27, %r12, %r27;\n\t" 
                    "mul.rn.f64 %r28, %r12, %r28;\n\t" 
                    "mul.rn.f64 %r13, %r12, %r13;\n\t" 
                    "mul.rn.f64 %r14, %r12, %r14;\n\t" 
                    "mul.rn.f64 %r15, %r12, %r15;\n\t" 
                    "mul.rn.f64 %r16, %r12, %r16;\n\t" 
                    "mul.rn.f64 %r17, %r12, %r17;\n\t" 
                    "mul.rn.f64 %r18, %r12, %r18;\n\t" 
                    "mul.rn.f64 %r19, %r12, %r19;\n\t" 
                    "mul.rn.f64 %r20, %r12, %r20;\n\t" 
                    "mul.rn.f64 %r21, %r12, %r21;\n\t" 
                    "mul.rn.f64 %r22, %r12, %r22;\n\t" 
                    "mul.rn.f64 %r23, %r12, %r23;\n\t" 
                    "mul.rn.f64 %r24, %r12, %r24;\n\t" 
                    "mul.rn.f64 %r25, %r12, %r25;\n\t" 
                    "mul.rn.f64 %r26, %r12, %r26;\n\t" 
                    "mul.rn.f64 %r27, %r12, %r27;\n\t" 
                    "mul.rn.f64 %r28, %r12, %r28;\n\t" 
                    "mul.rn.f64 %r13, %r12, %r13;\n\t" 
                    "mul.rn.f64 %r14, %r12, %r14;\n\t" 
                    "mul.rn.f64 %r15, %r12, %r15;\n\t" 
                    "mul.rn.f64 %r16, %r12, %r16;\n\t" 
                    "mul.rn.f64 %r17, %r12, %r17;\n\t" 
                    "mul.rn.f64 %r18, %r12, %r18;\n\t" 
                    "mul.rn.f64 %r19, %r12, %r19;\n\t" 
                    "mul.rn.f64 %r20, %r12, %r20;\n\t" 
                    "mul.rn.f64 %r21, %r12, %r21;\n\t" 
                    "mul.rn.f64 %r22, %r12, %r22;\n\t" 
                    "mul.rn.f64 %r23, %r12, %r23;\n\t" 
                    "mul.rn.f64 %r24, %r12, %r24;\n\t" 
                    "mul.rn.f64 %r25, %r12, %r25;\n\t" 
                    "mul.rn.f64 %r26, %r12, %r26;\n\t" 
                    "mul.rn.f64 %r27, %r12, %r27;\n\t" 
                    "mul.rn.f64 %r28, %r12, %r28;\n\t" 
                    );
        }
    }
    __syncthreads();

    //    if ((blockDim.x * blockIdx.x + threadIdx.x) == 0)
    *D = I1;

    __syncthreads();
}

void usage() {
    std::cout << "Usage ./binary <num_blocks> <num_threads_per_block> <iterations>" "threads active per warp" << std::endl;
}

int main(int argc, char **argv)
{
    if (argc != 5) {
        usage();
        exit(1);
    }

    int num_blocks = atoi(argv[1]);
    int num_threads_per_block = atoi(argv[2]);
    int iterations = atoi(argv[3]);
    int divergence = atoi(argv[4]);

//    h_A = new float(2.0);
//    h_B = new float(3.0);
//    h_C = new float(4.0);

//    cudaMalloc((void**)&d_A, sizeof(float));
//    cudaMalloc((void**)&d_B, sizeof(float));
//    cudaMalloc((void**)&d_C, sizeof(float));
    hipMalloc((void**)&d_res, sizeof(float));
  
//    cudaMemcpy(d_A, h_A, sizeof(float), cudaMemcpyHostToDevice);
//    cudaMemcpy(d_B, h_B, sizeof(float), cudaMemcpyHostToDevice);
//    cudaMemcpy(d_C, h_C, sizeof(float), cudaMemcpyHostToDevice);
     
    hipEvent_t start, stop;
    float time;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    hipProfilerStart();
    
//    compute<<<num_blocks, num_threads_per_block>>>(d_A, d_B, d_C, d_res, iterations);
    compute<<<num_blocks, num_threads_per_block>>>(d_res, iterations, divergence);

    hipProfilerStop();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&time, start, stop);

    std::cout << "GPU Elapsed Time = " << time << std::endl;
  
    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipDeviceSynchronize();

    hipMemcpy(h_res, d_res, sizeof(float), hipMemcpyDeviceToHost);

    return 0;
}
